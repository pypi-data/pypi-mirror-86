#include "hip/hip_runtime.h"
#include "custom_cuda_layers.h"

__global__ void dropout_kernel(const int N,
                               const float ratio,
                               float* out,
                               const float* Xdata,
                               uint8_t* mask,
                               std::pair<uint64_t, uint64_t> seed)
{
    const float scale = 1. / (1. - ratio);
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    hiprandStatePhilox4_32_10_t state;
    hiprand_init(seed.first, idx, seed.second, &state);

    CUDA_1D_KERNEL_LOOP(j, N / 4)
    {
        float4 rand = hiprand_uniform4(&state);
        uint8_t m[4];

        m[0] = (uint8_t)(rand.x > ratio);
        m[1] = (uint8_t)(rand.y > ratio);
        m[2] = (uint8_t)(rand.z > ratio);
        m[3] = (uint8_t)(rand.w > ratio);

        int i = j * 4;

        mask[i] = (uint8_t)m[0];
        mask[i + 1] = (uint8_t)m[1];
        mask[i + 2] = (uint8_t)m[2];
        mask[i + 3] = (uint8_t)m[3];

        out[i] = Xdata[i] * scale * m[0];
        out[i + 1] = Xdata[i + 1] * scale * m[1];
        out[i + 2] = Xdata[i + 2] * scale * m[2];
        out[i + 3] = Xdata[i + 3] * scale * m[3];
    }
}

__global__ void dropout_kernel(const int N,
                               const float ratio,
                               __half* out,
                               const __half* Xdata,
                               uint8_t* mask,
                               std::pair<uint64_t, uint64_t> seed)
{
    const float scale = 1. / (1. - ratio);

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    hiprandStatePhilox4_32_10_t state;
    hiprand_init(seed.first, idx, seed.second, &state);

#ifdef __STOCHASTIC_MODE__

    const __half2 h_scale = __float2half2_rn(scale);
    const float2* x_cast = reinterpret_cast<const float2*>(Xdata);
    float2* out_cast = reinterpret_cast<float2*>(out);
    uint32_t* mask_cast = reinterpret_cast<uint32_t*>(mask);

    uint32_t m_32;
    uint8_t* m = reinterpret_cast<uint8_t*>(&m_32);

    float2 result_f;
    __half2* result_h = reinterpret_cast<__half2*>(&result_f);
    __half2 mask_h[2];
    float2 mask_f[2];

    CUDA_1D_KERNEL_LOOP(j, N / 4)
    {
        float2 x_f = x_cast[j];
        __half2* x_h = reinterpret_cast<__half2*>(&x_f);

        float4 rand = hiprand_uniform4(&state);

        m[0] = (uint8_t)(rand.x > ratio);
        m[1] = (uint8_t)(rand.y > ratio);
        m[2] = (uint8_t)(rand.z > ratio);
        m[3] = (uint8_t)(rand.w > ratio);

        float* mask_f_data = &mask_f[0].x;
#pragma unroll
        for (int i = 0; i < 4; i++) mask_f_data[i] = (float)(m[i]);

        mask_h[0] = __float22half2_rn(mask_f[0]);
        mask_h[1] = __float22half2_rn(mask_f[1]);

        result_h[0] = x_h[0] * h_scale * mask_h[0];
        result_h[1] = x_h[1] * h_scale * mask_h[1];

        out_cast[j] = result_f;

        mask_cast[j] = m_32;
    }

#else

    CUDA_1D_KERNEL_LOOP(j, N / 4)
    {
        int i = j * 4;

        const __half2* vals_half = reinterpret_cast<const __half2*>(Xdata + i);
        float2 vals_half_f[2];
        vals_half_f[0] = __half22float2(vals_half[0]);
        vals_half_f[1] = __half22float2(vals_half[1]);

        uint8_t m[4];
        float4 rand = hiprand_uniform4(&state);
        m[0] = (uint8_t)(rand.x > ratio);
        m[1] = (uint8_t)(rand.y > ratio);
        m[2] = (uint8_t)(rand.z > ratio);
        m[3] = (uint8_t)(rand.w > ratio);

        out[i] = __float2half(vals_half_f[0].x * scale * m[0]);
        out[i + 1] = __float2half(vals_half_f[0].y * scale * m[1]);
        out[i + 2] = __float2half(vals_half_f[1].x * scale * m[2]);
        out[i + 3] = __float2half(vals_half_f[1].y * scale * m[3]);

        mask[i] = m[0];
        mask[i + 1] = m[1];
        mask[i + 2] = m[2];
        mask[i + 3] = m[3];
    }

#endif
}

__global__ void dropout_kernel_bwd(const int N,
                                   const float ratio,
                                   const float* Xdata,
                                   float* out,
                                   uint8_t* mask,
                                   std::pair<uint64_t, uint64_t> seed)
{
    const float scale = 1. / (1. - ratio);
    CUDA_1D_KERNEL_LOOP(j, N / 4)
    {
        int i = j * 4;

        out[i] = mask[i] ? Xdata[i] * scale : 0.0;
        out[i + 1] = mask[i + 1] ? Xdata[i + 1] * scale : 0.0;
        out[i + 2] = mask[i + 2] ? Xdata[i + 2] * scale : 0.0;
        out[i + 3] = mask[i + 3] ? Xdata[i + 3] * scale : 0.0;
    }
}

__global__ void dropout_kernel_bwd(const int N,
                                   const float ratio,
                                   const __half* Xdata,
                                   __half* out,
                                   uint8_t* mask,
                                   std::pair<uint64_t, uint64_t> seed)
{
    const float scale = 1. / (1. - ratio);

#ifdef __STOCHASTIC_MODE__

    const __half2 h_scale = __float2half2_rn(scale);

    const float2* x_cast = reinterpret_cast<const float2*>(Xdata);
    float2* out_cast = reinterpret_cast<float2*>(out);
    uint32_t* mask_cast = reinterpret_cast<uint32_t*>(mask);

    CUDA_1D_KERNEL_LOOP(j, N / 4)
    {
        float2 x_f = x_cast[j];
        __half2* x_h = reinterpret_cast<__half2*>(&x_f);

        uint8_t* m = reinterpret_cast<uint8_t*>(mask_cast + j);
        __half2 mask_h[2];
        float2 mask_f[2];

        float* mask_f_data = &mask_f[0].x;
#pragma unroll
        for (int i = 0; i < 4; i++) mask_f_data[i] = (float)(m[i]);

#pragma unroll
        for (int i = 0; i < 2; i++) mask_h[i] = __float22half2_rn(mask_f[i]);

        float2 result_f;
        __half2* result_h = reinterpret_cast<__half2*>(&result_f);

        result_h[0] = x_h[0] * h_scale * mask_h[0];
        result_h[1] = x_h[1] * h_scale * mask_h[1];

        out_cast[j] = result_f;
    }

#else

    const __half h_scale = __float2half(scale);
    const __half h_zero = __float2half(0.0);

    CUDA_1D_KERNEL_LOOP(j, N / 4)
    {
        int i = j * 4;

        const __half2* vals_half = reinterpret_cast<const __half2*>(Xdata + i);

        uint8_t* m = mask + i;

        float2 vals_half_f[2];

        vals_half_f[0] = __half22float2(vals_half[0]);
        vals_half_f[1] = __half22float2(vals_half[1]);

        out[i] = __float2half(vals_half_f[0].x * scale * m[0]);
        out[i + 1] = __float2half(vals_half_f[0].y * scale * m[1]);
        out[i + 2] = __float2half(vals_half_f[1].x * scale * m[2]);
        out[i + 3] = __float2half(vals_half_f[1].y * scale * m[3]);
    }

#endif
}

template <typename T>
void launch_dropout(T* out,
                    const T* vals,
                    uint8_t* mask,
                    int total_count,
                    int dim,
                    float ratio,
                    hipStream_t stream,
                    bool bwd)
{
    dim3 grid_dim = DS_GET_BLOCKS(total_count / 4);
    dim3 block_dim = DS_CUDA_NUM_THREADS;

    if (dim > 512) {
        block_dim.x >>= 1;
        grid_dim.x <<= 1;
    }
    uint64_t inc = total_count / grid_dim.x / block_dim.x;
    std::pair<uint64_t, uint64_t> seed = Context::Instance().IncrementOffset(inc);
    if (bwd)
        dropout_kernel_bwd<<<grid_dim, block_dim, 0, stream>>>(
            total_count, ratio, vals, out, mask, seed);
    else
        dropout_kernel<<<grid_dim, block_dim, 0, stream>>>(
            total_count, ratio, out, vals, mask, seed);
}

template void launch_dropout(float* out,
                             const float* vals,
                             uint8_t* mask,
                             int total_count,
                             int dim,
                             float ratio,
                             hipStream_t stream,
                             bool);
template void launch_dropout(__half* out,
                             const __half* vals,
                             uint8_t* mask,
                             int total_count,
                             int dim,
                             float ratio,
                             hipStream_t stream,
                             bool);

__global__ void dropout_grad_kernel(const int N, const float scale, float* Xdata, uint8_t* mask)
{
    CUDA_1D_KERNEL_LOOP(i, N) { Xdata[i] *= scale * mask[i]; }
}

__global__ void dropout_grad_kernel(const int N, const float scale, __half* Xdata, uint8_t* mask)
{
#ifdef __STOCHASTIC_MODE__

    const __half2 h_scale = __float2half2_rn(scale);
    float2* x_cast = reinterpret_cast<float2*>(Xdata);
    uint32_t* mask_cast = reinterpret_cast<uint32_t*>(mask);

    CUDA_1D_KERNEL_LOOP(j, N / 4)
    {
        uint8_t* m = reinterpret_cast<uint8_t*>(mask_cast + j);
        __half2 mask_h[2];
        float2 mask_f[2];

        float* mask_f_data = &mask_f[0].x;
#pragma unroll
        for (int i = 0; i < 4; i++) *(mask_f_data++) = (float)(m[i]);

        mask_h[0] = __float22half2_rn(mask_f[0]);
        mask_h[1] = __float22half2_rn(mask_f[1]);

        float2 x_data = x_cast[j];
        __half2* x_data_h = reinterpret_cast<__half2*>(&x_data);

        float2 result_f;
        __half2* result_h = reinterpret_cast<__half2*>(&result_f);

        result_h[0] = x_data_h[0] * h_scale * mask_h[0];
        result_h[1] = x_data_h[1] * h_scale * mask_h[1];

        x_cast[j] = result_f;
    }

#else

    CUDA_1D_KERNEL_LOOP(j, N / 2)
    {
        int i = j * 2;
        Xdata[i] = (__half)((float)Xdata[i] * scale * mask[i]);
        Xdata[i + 1] = (__half)((float)Xdata[i + 1] * scale * mask[i + 1]);
    }

#endif
}

template <typename T>
void launch_dropout_grad(T* vals, uint8_t* mask, int total_count, float ratio, hipStream_t stream)
{
    const float scale = 1. / (1. - ratio);
    dropout_grad_kernel<<<DS_GET_BLOCKS(total_count / 2), DS_CUDA_NUM_THREADS, 0, stream>>>(
        total_count, scale, vals, mask);
}

template void launch_dropout_grad(float* vals,
                                  uint8_t* mask,
                                  int total_count,
                                  float ratio,
                                  hipStream_t stream);
template void launch_dropout_grad(__half* vals,
                                  uint8_t* mask,
                                  int total_count,
                                  float ratio,
                                  hipStream_t stream);

__global__ void dropout_grad_kernel(const int N,
                                    const float scale,
                                    const float* Xdata,
                                    float* out,
                                    uint8_t* mask)
{
    CUDA_1D_KERNEL_LOOP(i, N) { out[i] = Xdata[i] * scale * mask[i]; }
}

__global__ void dropout_grad_kernel(const int N,
                                    const float scale,
                                    const __half* Xdata,
                                    __half* out,
                                    uint8_t* mask)
{
    CUDA_1D_KERNEL_LOOP(j, N / 2)
    {
        int i = j * 2;
        out[i] = (__half)((float)Xdata[i] * scale * mask[i]);
        out[i + 1] = (__half)((float)Xdata[i + 1] * scale * mask[i + 1]);
    }
}

template <typename T>
void launch_dropout_grad(T* vals_out,
                         const T* vals,
                         uint8_t* mask,
                         int total_count,
                         float ratio,
                         hipStream_t stream)
{
    const float scale = 1. / (1. - ratio);
    dropout_grad_kernel<<<DS_GET_BLOCKS(total_count / 2), DS_CUDA_NUM_THREADS, 0, stream>>>(
        total_count, scale, vals, vals_out, mask);
}
template void launch_dropout_grad(float*,
                                  const float* vals,
                                  uint8_t* mask,
                                  int total_count,
                                  float ratio,
                                  hipStream_t stream);
template void launch_dropout_grad(__half*,
                                  const __half* vals,
                                  uint8_t* mask,
                                  int total_count,
                                  float ratio,
                                  hipStream_t stream);

__global__ void dropout_kernel(const int dim,
                               const float ratio,
                               const float* bias,
                               float* Xdata,
                               uint8_t* mask,
                               std::pair<uint64_t, uint64_t> seed)
{
    const float scale = 1. / (1. - ratio);
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    hiprandStatePhilox4_32_10_t state;
    hiprand_init(seed.first, idx, seed.second, &state);

    float4* Xdata_cast = reinterpret_cast<float4*>(Xdata);
    const float4* bias_cast = reinterpret_cast<const float4*>(bias);

    {
        float4 rand = hiprand_uniform4(&state);
        uint8_t m[4];

        m[0] = (uint8_t)(rand.x > ratio);
        m[1] = (uint8_t)(rand.y > ratio);
        m[2] = (uint8_t)(rand.z > ratio);
        m[3] = (uint8_t)(rand.w > ratio);

        int i = blockIdx.x * dim + tid * 4;

        float4 x_data = Xdata_cast[idx];
        float4 b_data = bias_cast[tid];

        x_data.x += b_data.x;
        x_data.y += b_data.y;
        x_data.z += b_data.z;
        x_data.w += b_data.w;

        x_data.x = x_data.x * scale * m[0];
        x_data.y = x_data.y * scale * m[1];
        x_data.z = x_data.z * scale * m[2];
        x_data.w = x_data.w * scale * m[3];

        mask[i] = (uint8_t)m[0];
        mask[i + 1] = (uint8_t)m[1];
        mask[i + 2] = (uint8_t)m[2];
        mask[i + 3] = (uint8_t)m[3];

        Xdata_cast[idx] = x_data;
    }
}

__global__ void dropout_kernel(const int dim,
                               const float ratio,
                               const __half* bias,
                               __half* Xdata,
                               uint8_t* mask,
                               std::pair<uint64_t, uint64_t> seed)
{
    const float scale = 1. / (1. - ratio);
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    hiprandStatePhilox4_32_10_t state;
    hiprand_init(seed.first, idx, seed.second, &state);

    float2* Xdata_cast = reinterpret_cast<float2*>(Xdata);
    const float2* bias_cast = reinterpret_cast<const float2*>(bias);

    {
        int i = blockIdx.x * dim + tid * 4;

        float4 rand = hiprand_uniform4(&state);

        float2 data_f;
        __half2* data_h = reinterpret_cast<__half2*>(&data_f);

        float2 bias_f;
        __half2* bias_h = reinterpret_cast<__half2*>(&bias_f);

        data_f = Xdata_cast[idx];
        bias_f = bias_cast[tid];

        float2 data_h_0 = __half22float2(data_h[0]);
        float2 data_h_1 = __half22float2(data_h[1]);

        float2 bias_h_0 = __half22float2(bias_h[0]);
        float2 bias_h_1 = __half22float2(bias_h[1]);

        data_h_0.x += bias_h_0.x;
        data_h_0.y += bias_h_0.y;
        data_h_1.x += bias_h_1.x;
        data_h_1.y += bias_h_1.y;

        uint8_t m[4];  // = mask + i;

        m[0] = (uint8_t)(rand.x > ratio);
        m[1] = (uint8_t)(rand.y > ratio);
        m[2] = (uint8_t)(rand.z > ratio);
        m[3] = (uint8_t)(rand.w > ratio);

        data_h_0.x = __float2half(data_h_0.x * scale * m[0]);
        data_h_0.y = __float2half(data_h_0.y * scale * m[1]);
        data_h_1.x = __float2half(data_h_1.x * scale * m[2]);
        data_h_1.y = __float2half(data_h_1.y * scale * m[3]);

        float2 result_f;
        __half2* result_h = reinterpret_cast<__half2*>(&result_f);

        result_h[0] = __float22half2_rn(data_h_0);
        result_h[1] = __float22half2_rn(data_h_1);

        Xdata_cast[idx] = result_f;

        mask[i] = m[0];
        mask[i + 1] = m[1];
        mask[i + 2] = m[2];
        mask[i + 3] = m[3];
    }
}

template <typename T>
void launch_dropout(T* out,
                    const T* bias,
                    uint8_t* mask,
                    int batch,
                    int dim,
                    float ratio,
                    hipStream_t stream)
{
    dim3 grid_dim(batch);     // DS_GET_BLOCKS(total_count/4);
    dim3 block_dim(dim / 4);  // DS_CUDA_NUM_THREADS;

    uint64_t inc = (batch * dim) / grid_dim.x / block_dim.x;
    std::pair<uint64_t, uint64_t> seed = Context::Instance().IncrementOffset(inc);

    dropout_kernel<<<grid_dim, block_dim, 0, stream>>>(dim, ratio, bias, out, mask, seed);
}

template void launch_dropout(float*,
                             const float* bias,
                             uint8_t* mask,
                             int batch,
                             int dim,
                             float ratio,
                             hipStream_t stream);
template void launch_dropout(__half*,
                             const __half* bias,
                             uint8_t* mask,
                             int batch,
                             int dim,
                             float ratio,
                             hipStream_t stream);

__global__ void dropout_kernel(const int dim,
                               const float ratio,
                               const float* input,
                               const float* residual,
                               const float* bias,
                               float* out,
                               uint8_t* mask,
                               std::pair<uint64_t, uint64_t> seed)
{
    const float scale = 1. / (1. - ratio);
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    hiprandStatePhilox4_32_10_t state;
    hiprand_init(seed.first, idx, seed.second, &state);

    float4* out_cast = reinterpret_cast<float4*>(out);
    const float4* bias_cast = reinterpret_cast<const float4*>(bias);
    const float4* residual_cast = reinterpret_cast<const float4*>(residual);
    const float4* input_cast = reinterpret_cast<const float4*>(input);

    {
        float4 rand = hiprand_uniform4(&state);
        uint8_t m[4];
        m[0] = (uint8_t)(rand.x > ratio);
        m[1] = (uint8_t)(rand.y > ratio);
        m[2] = (uint8_t)(rand.z > ratio);
        m[3] = (uint8_t)(rand.w > ratio);

        // int bid = k * blockDim.x + tid;
        int i = blockIdx.x * dim + tid * 4;

        float4 out_data = out_cast[idx];
        float4 b_data = bias_cast[tid];
        float4 res_data = residual_cast[idx];
        float4 inp_data = input_cast[idx];

        out_data.x = (b_data.x + inp_data.x);
        out_data.y = (b_data.y + inp_data.y);
        out_data.z = (b_data.z + inp_data.z);
        out_data.w = (b_data.w + inp_data.w);

        out_data.x = out_data.x * scale * m[0];
        out_data.y = out_data.y * scale * m[1];
        out_data.z = out_data.z * scale * m[2];
        out_data.w = out_data.w * scale * m[3];

        out_data.x += res_data.x;
        out_data.y += res_data.y;
        out_data.z += res_data.z;
        out_data.w += res_data.w;

        mask[i] = m[0];
        mask[i + 1] = m[1];
        mask[i + 2] = m[2];
        mask[i + 3] = m[3];

        out_cast[idx] = out_data;
    }
}

__global__ void dropout_kernel(const int dim,
                               const float ratio,
                               const __half* input,
                               const __half* residual,
                               const __half* bias,
                               __half* out,
                               uint8_t* mask,
                               std::pair<uint64_t, uint64_t> seed)
{
    const float scale = 1. / (1. - ratio);
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    hiprandStatePhilox4_32_10_t state;
    hiprand_init(seed.first, idx, seed.second, &state);

    float2* out_cast = reinterpret_cast<float2*>(out);
    const float2* bias_cast = reinterpret_cast<const float2*>(bias);
    const float2* residual_cast = reinterpret_cast<const float2*>(residual);
    const float2* input_cast = reinterpret_cast<const float2*>(input);

    {
        int i = blockIdx.x * dim + tid * 4;

        float4 rand = hiprand_uniform4(&state);

        float2 data_f;
        __half2* data_h = reinterpret_cast<__half2*>(&data_f);

        float2 bias_f;
        __half2* bias_h = reinterpret_cast<__half2*>(&bias_f);

        float2 residual_f;
        __half2* residual_h = reinterpret_cast<__half2*>(&residual_f);

        float2 input_f;
        __half2* input_h = reinterpret_cast<__half2*>(&input_f);

        data_f = out_cast[idx];
        bias_f = bias_cast[tid];
        residual_f = residual_cast[idx];
        input_f = input_cast[idx];

        float2 data_h_0 = __half22float2(data_h[0]);
        float2 data_h_1 = __half22float2(data_h[1]);

        float2 bias_h_0 = __half22float2(bias_h[0]);
        float2 bias_h_1 = __half22float2(bias_h[1]);

        float2 residual_h_0 = __half22float2(residual_h[0]);
        float2 residual_h_1 = __half22float2(residual_h[1]);

        float2 input_h_0 = __half22float2(input_h[0]);
        float2 input_h_1 = __half22float2(input_h[1]);

        data_h_0.x = (bias_h_0.x + input_h_0.x);
        data_h_0.y = (bias_h_0.y + input_h_0.y);
        data_h_1.x = (bias_h_1.x + input_h_1.x);
        data_h_1.y = (bias_h_1.y + input_h_1.y);

        uint8_t m[4];  // = mask + i;

        m[0] = (uint8_t)(rand.x > ratio);
        m[1] = (uint8_t)(rand.y > ratio);
        m[2] = (uint8_t)(rand.z > ratio);
        m[3] = (uint8_t)(rand.w > ratio);

        data_h_0.x = __float2half(data_h_0.x * scale * m[0]);
        data_h_0.y = __float2half(data_h_0.y * scale * m[1]);
        data_h_1.x = __float2half(data_h_1.x * scale * m[2]);
        data_h_1.y = __float2half(data_h_1.y * scale * m[3]);

        data_h_0.x += residual_h_0.x;
        data_h_0.y += residual_h_0.y;
        data_h_1.x += residual_h_1.x;
        data_h_1.y += residual_h_1.y;

        float2 result_f;
        __half2* result_h = reinterpret_cast<__half2*>(&result_f);

        result_h[0] = __float22half2_rn(data_h_0);
        result_h[1] = __float22half2_rn(data_h_1);

        out_cast[idx] = result_f;

        mask[i] = m[0];
        mask[i + 1] = m[1];
        mask[i + 2] = m[2];
        mask[i + 3] = m[3];
    }
}

template <typename T>
void launch_dropout(T* out,
                    const T* input,
                    const T* residual,
                    const T* bias,
                    uint8_t* mask,
                    int batch,
                    int dim,
                    float ratio,
                    hipStream_t stream)
{
    dim3 grid_dim(batch);     // DS_GET_BLOCKS(total_count/4);
    dim3 block_dim(dim / 4);  // DS_CUDA_NUM_THREADS;

    uint64_t inc = (batch * dim) / grid_dim.x / block_dim.x;
    std::pair<uint64_t, uint64_t> seed = Context::Instance().IncrementOffset(inc);

    dropout_kernel<<<grid_dim, block_dim, 0, stream>>>(
        dim, ratio, input, residual, bias, out, mask, seed);
}

template void launch_dropout(float*,
                             const float*,
                             const float* residual,
                             const float* bias,
                             uint8_t* mask,
                             int batch,
                             int dim,
                             float ratio,
                             hipStream_t stream);
template void launch_dropout(__half*,
                             const __half*,
                             const __half* residual,
                             const __half* bias,
                             uint8_t* mask,
                             int batch,
                             int dim,
                             float ratio,
                             hipStream_t stream);

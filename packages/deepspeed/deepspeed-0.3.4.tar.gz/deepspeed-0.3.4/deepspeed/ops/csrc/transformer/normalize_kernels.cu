#include "hip/hip_runtime.h"
#include "custom_cuda_layers.h"

namespace cg = cooperative_groups;

/*
Fused bias add, residual (elementwise) add, and normalization layer.

Unlike the GELU, which doesn't require template parameters, this layer does since it
does rely fairly heavily on unrolling loops. Currently, I exclude bounds checks and
assume that the number of elements is a multiple of a power of 2. Default behavior
for our purposes uses 256 threads for floats, and 128 threads for __half. This restriction
is a result of using the shift parameter to perform the minimum number of register file
shuffles necessary, which requires the number of threads in the secondary reduction to
be 1, 2, 4, 8, 16, or 32. The number of threads here corresponds to the number of complete
warps in the threadblock.

For FP16, this kernel does not promote to FP32 in order to utilize the 2x throughput for
__half2 instructions, and avoid the conversion overhead (1/8 of __hal2 arithmetic).

For specific launch constraints, see the launch functions.
*/

template <int row_stride, int iterations>
__global__ void fused_bias_residual_layer_norm(float* vals,
                                               const float* residual,
                                               const float* gamma,
                                               const float* beta,
                                               float epsilon,
                                               bool preLayerNorm,
                                               bool training,
                                               float* vars,
                                               float* means)
{
    constexpr int iteration_stride = row_stride / iterations;

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<32> g = cg::tiled_partition<32>(b);

    int row = blockIdx.x;
    int id = threadIdx.x;
    int gid = id / 32;

    float vals_arr[iterations];
    __shared__ float shr[iteration_stride >> 5];

    float sum = 0.f;
#pragma unroll
    for (int i = 0; i < iterations; i++) {
        vals_arr[i] = residual[row * row_stride + i * iteration_stride + id];
        sum += vals_arr[i];
    }

    for (int i = 1; i < 32; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) shr[gid] = sum;

    b.sync();

    if (g.thread_rank() < (iteration_stride >> 5)) sum = shr[g.thread_rank()];

#if !defined(__STOCHASTIC_MODE__) || __CUDA_ARCH__ < 700
    b.sync();
#endif

    for (int i = 1; i < (iteration_stride >> 5); i *= 2) { sum += g.shfl_down(sum, i); }

    sum = g.shfl(sum, 0);
    float mean = sum / row_stride;
    if (training)
        if (g.thread_rank() == 0) means[row] = mean;
    float variance = 0.f;
    for (int i = 0; i < iterations; i++) {
        variance += (vals_arr[i] - mean) * (vals_arr[i] - mean);
    }

    for (int i = 1; i < 32; i *= 2) { variance += g.shfl_down(variance, i); }

    if (g.thread_rank() == 0) shr[gid] = variance;

    b.sync();

    if (g.thread_rank() < (iteration_stride >> 5)) variance = shr[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    b.sync();
#endif

    for (int i = 1; i < (iteration_stride >> 5); i *= 2) { variance += g.shfl_down(variance, i); }
    variance = g.shfl(variance, 0);
    variance /= row_stride;
    variance += epsilon;
    if (training)
        if (g.thread_rank() == 0) vars[row] = variance;

    for (int i = 0; i < iterations; i++) {
        vals_arr[i] = (vals_arr[i] - mean) * rsqrtf(variance);
        vals_arr[i] =
            vals_arr[i] * gamma[i * iteration_stride + id] + beta[i * iteration_stride + id];
        vals[row * row_stride + i * iteration_stride + id] = vals_arr[i];
    }
}

template <int row_stride, int iterations>
__global__ void fused_bias_residual_layer_norm(__half* vals,
                                               const __half* residual,
                                               const __half* gamma,
                                               const __half* beta,
                                               float epsilon,
                                               bool preLayerNorm,
                                               bool training,
                                               __half* vars,
                                               __half* means)
{
#if __CUDA_ARCH__ >= 700
    constexpr int iteration_stride = row_stride / iterations;

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<32> g = cg::tiled_partition<32>(b);

    int row = blockIdx.x;
    int id = threadIdx.x;
    int gid = id >> 5;

    __half2 vals_arr[iterations];
    float2 vals_f[iterations];
    __shared__ float shr[iteration_stride >> 5];

    __half2* vals_cast = reinterpret_cast<__half2*>(vals);
    const __half2* residual_cast = reinterpret_cast<const __half2*>(residual);

    float sum = 0.f;
#pragma unroll
    for (int i = 0; i < iterations; i++) {
        vals_f[i] = __half22float2(residual_cast[row * row_stride + i * iteration_stride + id]);
        sum += vals_f[i].x;
        sum += vals_f[i].y;
    }

    for (int i = 1; i < 32; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) shr[gid] = sum;

    b.sync();

    if (g.thread_rank() < (iteration_stride >> 5)) sum = shr[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    b.sync();
#endif

    for (int i = 1; i < (iteration_stride >> 5); i *= 2) { sum += g.shfl_down(sum, i); }
    sum = g.shfl(sum, 0);
    float mean = sum / (row_stride * 2);

    float variance = 0.f;
    for (int i = 0; i < iterations; i++) {
        variance += (vals_f[i].x - mean) * (vals_f[i].x - mean);
        variance += (vals_f[i].y - mean) * (vals_f[i].y - mean);
    }

    for (int i = 1; i < 32; i *= 2) { variance += g.shfl_down(variance, i); }

    if (g.thread_rank() == 0) shr[gid] = variance;

    b.sync();

    if (g.thread_rank() < (iteration_stride >> 5)) variance = shr[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    b.sync();
#endif

    for (int i = 1; i < (iteration_stride >> 5); i *= 2) { variance += g.shfl_down(variance, i); }
    variance = g.shfl(variance, 0);
    variance /= (row_stride * 2);
    variance += epsilon;

    __half2 mean_h = __float2half2_rn(mean);
    __half2 variance_h = __float2half2_rn(variance);
    const __half2* gamma_cast = reinterpret_cast<const __half2*>(gamma);
    const __half2* beta_cast = reinterpret_cast<const __half2*>(beta);

    if (training && g.thread_rank() == 0) {
        vars[row] = __float2half(variance);
        means[row] = __float2half(mean);
    }

    for (int i = 0; i < iterations; i++) {
        vals_arr[i] = __float22half2_rn(vals_f[i]);
        vals_arr[i] = (vals_arr[i] - mean_h) * h2rsqrt(variance_h);
        vals_arr[i] = vals_arr[i] * gamma_cast[i * iteration_stride + id] +
                      beta_cast[i * iteration_stride + id];
        vals_cast[row * row_stride + i * iteration_stride + id] = vals_arr[i];
    }
#endif
}

template <typename T>
void launch_bias_residual_layer_norm(T* vals,
                                     const T* residual,
                                     const T* gamma,
                                     const T* beta,
                                     float epsilon,
                                     int batch_size,
                                     int hidden_dim,
                                     hipStream_t stream,
                                     bool preLayerNorm,
                                     bool training,
                                     T* vars,
                                     T* means);

template <>
void launch_bias_residual_layer_norm<float>(float* vals,
                                            const float* residual,
                                            const float* gamma,
                                            const float* beta,
                                            float epsilon,
                                            int batch_size,
                                            int hidden_dim,
                                            hipStream_t stream,
                                            bool preLayerNorm,
                                            bool training,
                                            float* vars,
                                            float* means)
{
    constexpr int threads = THREADS;

    dim3 grid_dim(batch_size);

    dim3 block_dim(threads);

    // There are some limitations to call below functions, now just enumerate the situations.
    if (hidden_dim == 768)
        fused_bias_residual_layer_norm<768, 3><<<grid_dim, block_dim, 0, stream>>>(
            vals, residual, gamma, beta, epsilon, preLayerNorm, training, vars, means);
    else if (hidden_dim == 512)
        fused_bias_residual_layer_norm<512, 2><<<grid_dim, block_dim, 0, stream>>>(
            vals, residual, gamma, beta, epsilon, preLayerNorm, training, vars, means);
    else if (hidden_dim == 1024)
        fused_bias_residual_layer_norm<1024, 4><<<grid_dim, block_dim, 0, stream>>>(
            vals, residual, gamma, beta, epsilon, preLayerNorm, training, vars, means);
    else if (hidden_dim == 1536)
        fused_bias_residual_layer_norm<1536, 6><<<grid_dim, block_dim, 0, stream>>>(
            vals, residual, gamma, beta, epsilon, preLayerNorm, training, vars, means);
    else if (hidden_dim == 2048)
        fused_bias_residual_layer_norm<2048, 8><<<grid_dim, block_dim, 0, stream>>>(
            vals, residual, gamma, beta, epsilon, preLayerNorm, training, vars, means);
    else if (hidden_dim == 2560)
        fused_bias_residual_layer_norm<2560, 10><<<grid_dim, block_dim, 0, stream>>>(
            vals, residual, gamma, beta, epsilon, preLayerNorm, training, vars, means);
    else
        throw std::runtime_error("Unsupport hidden_dim.");
}

template <>
void launch_bias_residual_layer_norm<__half>(__half* vals,
                                             const __half* residual,
                                             const __half* gamma,
                                             const __half* beta,
                                             float epsilon,
                                             int batch_size,
                                             int hidden_dim,
                                             hipStream_t stream,
                                             bool preLayerNorm,
                                             bool training,
                                             __half* vars,
                                             __half* means)
{
    constexpr int threads = 128;

    dim3 grid_dim(batch_size);
    dim3 block_dim(threads);

    // There are some limitations to call below functions, now just enumerate the situations.
    if (hidden_dim == 768)
        fused_bias_residual_layer_norm<384, 3><<<grid_dim, block_dim, 0, stream>>>(
            vals, residual, gamma, beta, epsilon, preLayerNorm, training, vars, means);
    else if (hidden_dim == 512)
        fused_bias_residual_layer_norm<256, 2><<<grid_dim, block_dim, 0, stream>>>(
            vals, residual, gamma, beta, epsilon, preLayerNorm, training, vars, means);
    else if (hidden_dim == 1024)
        fused_bias_residual_layer_norm<512, 4><<<grid_dim, block_dim, 0, stream>>>(
            vals, residual, gamma, beta, epsilon, preLayerNorm, training, vars, means);
    else if (hidden_dim == 1536)
        fused_bias_residual_layer_norm<768, 6><<<grid_dim, block_dim, 0, stream>>>(
            vals, residual, gamma, beta, epsilon, preLayerNorm, training, vars, means);
    else if (hidden_dim == 2048)
        fused_bias_residual_layer_norm<1024, 8><<<grid_dim, block_dim, 0, stream>>>(
            vals, residual, gamma, beta, epsilon, preLayerNorm, training, vars, means);
    else if (hidden_dim == 2560)
        fused_bias_residual_layer_norm<1280, 10><<<grid_dim, block_dim, 0, stream>>>(
            vals, residual, gamma, beta, epsilon, preLayerNorm, training, vars, means);
    else
        throw std::runtime_error("Unsupport hidden_dim.");
}

template <int row_stride, int iterations>
__global__ void fused_bias_residual_layer_norm(float* vals,
                                               const float* residual,
                                               const float* gamma,
                                               const float* beta,
                                               float epsilon,
                                               bool preLayerNorm,
                                               bool training,
                                               float* vars)
{
    constexpr int iteration_stride = row_stride / iterations;

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<32> g = cg::tiled_partition<32>(b);

    int row = blockIdx.x;
    int id = threadIdx.x;
    int gid = id / 32;

    float vals_arr[iterations];
    __shared__ float shr[iteration_stride >> 5];

    float sum = 0.f;
#pragma unroll
    for (int i = 0; i < iterations; i++) {
        vals_arr[i] = residual[row * row_stride + i * iteration_stride + id];
        sum += vals_arr[i];
    }

    for (int i = 1; i < 32; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) shr[gid] = sum;

    b.sync();

    if (g.thread_rank() < (iteration_stride >> 5)) sum = shr[g.thread_rank()];

#if !defined(__STOCHASTIC_MODE__) || __CUDA_ARCH__ < 700
    b.sync();
#endif

    for (int i = 1; i < (iteration_stride >> 5); i *= 2) { sum += g.shfl_down(sum, i); }

    sum = g.shfl(sum, 0);
    float mean = sum / row_stride;
    float variance = 0.f;
    for (int i = 0; i < iterations; i++) {
        variance += (vals_arr[i] - mean) * (vals_arr[i] - mean);
    }

    for (int i = 1; i < 32; i *= 2) { variance += g.shfl_down(variance, i); }

    if (g.thread_rank() == 0) shr[gid] = variance;

    b.sync();

    if (g.thread_rank() < (iteration_stride >> 5)) variance = shr[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    b.sync();
#endif

    for (int i = 1; i < (iteration_stride >> 5); i *= 2) { variance += g.shfl_down(variance, i); }
    variance = g.shfl(variance, 0);
    variance /= row_stride;
    variance += epsilon;
    if (training)
        if (g.thread_rank() == 0) vars[row] = variance;

    for (int i = 0; i < iterations; i++) {
        vals_arr[i] = (vals_arr[i] - mean) * rsqrtf(variance);
        vals_arr[i] =
            vals_arr[i] * gamma[i * iteration_stride + id] + beta[i * iteration_stride + id];
        vals[row * row_stride + i * iteration_stride + id] = vals_arr[i];
    }
}

template <int row_stride, int iterations>
__global__ void fused_bias_residual_layer_norm(__half* vals,
                                               const __half* residual,
                                               const __half* gamma,
                                               const __half* beta,
                                               float epsilon,
                                               bool preLayerNorm,
                                               bool training,
                                               __half* vars)
{
#if __CUDA_ARCH__ >= 700
    constexpr int iteration_stride = row_stride / iterations;

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<32> g = cg::tiled_partition<32>(b);

    int row = blockIdx.x;
    int id = threadIdx.x;
    int gid = id >> 5;

    __half2 vals_arr[iterations];
    float2 vals_f[iterations];
    __shared__ float shr[iteration_stride >> 5];

    __half2* vals_cast = reinterpret_cast<__half2*>(vals);
    const __half2* residual_cast = reinterpret_cast<const __half2*>(residual);

    float sum = 0.f;
#pragma unroll
    for (int i = 0; i < iterations; i++) {
        vals_f[i] = __half22float2(residual_cast[row * row_stride + i * iteration_stride + id]);
        sum += vals_f[i].x;
        sum += vals_f[i].y;
    }

    for (int i = 1; i < 32; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) shr[gid] = sum;

    b.sync();

    if (g.thread_rank() < (iteration_stride >> 5)) sum = shr[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    b.sync();
#endif

    for (int i = 1; i < (iteration_stride >> 5); i *= 2) { sum += g.shfl_down(sum, i); }
    sum = g.shfl(sum, 0);
    float mean = sum / (row_stride * 2);

    float variance = 0.f;
    for (int i = 0; i < iterations; i++) {
        variance += (vals_f[i].x - mean) * (vals_f[i].x - mean);
        variance += (vals_f[i].y - mean) * (vals_f[i].y - mean);
    }

    for (int i = 1; i < 32; i *= 2) { variance += g.shfl_down(variance, i); }

    if (g.thread_rank() == 0) shr[gid] = variance;

    b.sync();

    if (g.thread_rank() < (iteration_stride >> 5)) variance = shr[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    b.sync();
#endif

    for (int i = 1; i < (iteration_stride >> 5); i *= 2) { variance += g.shfl_down(variance, i); }
    variance = g.shfl(variance, 0);
    variance /= (row_stride * 2);
    variance += epsilon;

    __half2 mean_h = __float2half2_rn(mean);
    __half2 variance_h = __float2half2_rn(variance);
    const __half2* gamma_cast = reinterpret_cast<const __half2*>(gamma);
    const __half2* beta_cast = reinterpret_cast<const __half2*>(beta);

    if (training && g.thread_rank() == 0) vars[row] = __float2half(variance);

    for (int i = 0; i < iterations; i++) {
        vals_arr[i] = __float22half2_rn(vals_f[i]);
        vals_arr[i] = (vals_arr[i] - mean_h) * h2rsqrt(variance_h);
        vals_arr[i] = vals_arr[i] * gamma_cast[i * iteration_stride + id] +
                      beta_cast[i * iteration_stride + id];
        vals_cast[row * row_stride + i * iteration_stride + id] = vals_arr[i];
    }
#endif
}

template <typename T>
void launch_bias_residual_layer_norm(T* vals,
                                     const T* residual,
                                     const T* gamma,
                                     const T* beta,
                                     float epsilon,
                                     int batch_size,
                                     int hidden_dim,
                                     hipStream_t stream,
                                     bool preLayerNorm,
                                     bool training,
                                     T* vars);

/*
To tune this launch the following restrictions must be met:

For float:
row_stride == hidden_size
threads * iterations == row_stride
threads is in [32, 64, 128, 256, 512, 1024]

For half:
row_stride == hidden_size / 2
threads * iterations == row_stride
threads is in [32, 64, 128, 256, 512, 1024]

*/

template <>
void launch_bias_residual_layer_norm<float>(float* vals,
                                            const float* residual,
                                            const float* gamma,
                                            const float* beta,
                                            float epsilon,
                                            int batch_size,
                                            int hidden_dim,
                                            hipStream_t stream,
                                            bool preLayerNorm,
                                            bool training,
                                            float* vars)
{
    constexpr int threads = THREADS;

    dim3 grid_dim(batch_size);

    dim3 block_dim(threads);

    // There are some limitations to call below functions, now just enumerate the situations.
    if (hidden_dim == 768)
        fused_bias_residual_layer_norm<768, 3><<<grid_dim, block_dim, 0, stream>>>(
            vals, residual, gamma, beta, epsilon, preLayerNorm, training, vars);
    else if (hidden_dim == 512)
        fused_bias_residual_layer_norm<512, 2><<<grid_dim, block_dim, 0, stream>>>(
            vals, residual, gamma, beta, epsilon, preLayerNorm, training, vars);
    else if (hidden_dim == 1024)
        fused_bias_residual_layer_norm<1024, 4><<<grid_dim, block_dim, 0, stream>>>(
            vals, residual, gamma, beta, epsilon, preLayerNorm, training, vars);
    else if (hidden_dim == 1536)
        fused_bias_residual_layer_norm<1536, 6><<<grid_dim, block_dim, 0, stream>>>(
            vals, residual, gamma, beta, epsilon, preLayerNorm, training, vars);
    else if (hidden_dim == 2048)
        fused_bias_residual_layer_norm<2048, 8><<<grid_dim, block_dim, 0, stream>>>(
            vals, residual, gamma, beta, epsilon, preLayerNorm, training, vars);
    else if (hidden_dim == 2560)
        fused_bias_residual_layer_norm<2560, 10><<<grid_dim, block_dim, 0, stream>>>(
            vals, residual, gamma, beta, epsilon, preLayerNorm, training, vars);
    else
        throw std::runtime_error("Unsupport hidden_dim.");
}

template <>
void launch_bias_residual_layer_norm<__half>(__half* vals,
                                             const __half* residual,
                                             const __half* gamma,
                                             const __half* beta,
                                             float epsilon,
                                             int batch_size,
                                             int hidden_dim,
                                             hipStream_t stream,
                                             bool preLayerNorm,
                                             bool training,
                                             __half* vars)
{
    constexpr int threads = 128;

    dim3 grid_dim(batch_size);
    dim3 block_dim(threads);

    // There are some limitations to call below functions, now just enumerate the situations.
    if (hidden_dim == 768)
        fused_bias_residual_layer_norm<384, 3><<<grid_dim, block_dim, 0, stream>>>(
            vals, residual, gamma, beta, epsilon, preLayerNorm, training, vars);
    else if (hidden_dim == 512)
        fused_bias_residual_layer_norm<256, 2><<<grid_dim, block_dim, 0, stream>>>(
            vals, residual, gamma, beta, epsilon, preLayerNorm, training, vars);
    else if (hidden_dim == 1024)
        fused_bias_residual_layer_norm<512, 4><<<grid_dim, block_dim, 0, stream>>>(
            vals, residual, gamma, beta, epsilon, preLayerNorm, training, vars);
    else if (hidden_dim == 1536)
        fused_bias_residual_layer_norm<768, 6><<<grid_dim, block_dim, 0, stream>>>(
            vals, residual, gamma, beta, epsilon, preLayerNorm, training, vars);
    else if (hidden_dim == 2048)
        fused_bias_residual_layer_norm<1024, 8><<<grid_dim, block_dim, 0, stream>>>(
            vals, residual, gamma, beta, epsilon, preLayerNorm, training, vars);
    else if (hidden_dim == 2560)
        fused_bias_residual_layer_norm<1280, 10><<<grid_dim, block_dim, 0, stream>>>(
            vals, residual, gamma, beta, epsilon, preLayerNorm, training, vars);
    else
        throw std::runtime_error("Unsupport hidden_dim.");
}

/* Normalize Gamma & Betta gradients
 * Compute gradients using either X_hat or
 * normalize input (invertible).
 * Combine transpose with gradients computation.
 */

template <typename T>
__global__ void LayerNormBackward1(const T* __restrict__ out_grad,
                                   const T* __restrict__ vals_hat,
                                   const T* __restrict__ gamma,
                                   const T* __restrict__ betta,
                                   T* __restrict__ gamma_grad,
                                   T* __restrict__ betta_grad,
                                   int rows,
                                   int width,
                                   bool invertible)
{
    __shared__ float betta_buffer[TILE_DIM][TILE_DIM + 1];
    __shared__ float gamma_buffer[TILE_DIM][TILE_DIM + 1];

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<TILE_DIM> g = cg::tiled_partition<TILE_DIM>(b);

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = threadIdx.y * width + idx;
    int y_stride = width * TILE_DIM;

    int pos = blockIdx.x * TILE_DIM + threadIdx.y;
    float betta_reg = (invertible ? (float)betta[pos] : 0.0f);
    float gamma_reg = (float)gamma[pos];

    // Loop across matrix height
    float betta_tmp = 0;
    float gamma_tmp = 0;
    for (int r = threadIdx.y; r < rows; r += TILE_DIM) {
        float grad = (float)out_grad[offset];
        float val = (invertible ? ((float)vals_hat[offset] - betta_reg) / gamma_reg
                                : (float)vals_hat[offset]);
        betta_tmp += grad;
        gamma_tmp += (val * grad);

        offset += y_stride;
    }

    betta_buffer[threadIdx.x][threadIdx.y] = betta_tmp;
    gamma_buffer[threadIdx.x][threadIdx.y] = gamma_tmp;

    __syncthreads();

    // Sum the shared buffer.
    float s1 = betta_buffer[threadIdx.y][threadIdx.x];
    float s2 = gamma_buffer[threadIdx.y][threadIdx.x];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < TILE_DIM; i <<= 1) {
        s1 += g.shfl_down(s1, i);
        s2 += g.shfl_down(s2, i);
    }

    if (threadIdx.x == 0) {
        betta_grad[pos] = s1;
        gamma_grad[pos] = s2;
    }
}

/* Normalize Gamma & Betta gradients
 * Compute gradients using the input to
 * the normalize.
 * Combine transpose with gradients computation.
 */

template <typename T>
__global__ void LayerNormBackward1(const T* __restrict__ out_grad,
                                   const T* __restrict__ X_data,
                                   const T* __restrict__ vars,
                                   const T* __restrict__ means,
                                   T* __restrict__ gamma_grad,
                                   T* __restrict__ betta_grad,
                                   int rows,
                                   int width)
{
    __shared__ float betta_buffer[TILE_DIM][TILE_DIM + 1];
    __shared__ float gamma_buffer[TILE_DIM][TILE_DIM + 1];

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<TILE_DIM> g = cg::tiled_partition<TILE_DIM>(b);

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = threadIdx.y * width + idx;
    int y_stride = width * TILE_DIM;

    int pos = blockIdx.x * TILE_DIM + threadIdx.y;
    // Loop across matrix height

    float betta_tmp = 0;
    float gamma_tmp = 0;
    for (int r = threadIdx.y; r < rows; r += TILE_DIM) {
        float grad = (float)out_grad[offset];
        float val = (float)X_data[offset];
        val = (val - (float)means[r]) * rsqrtf((float)vars[r]);
        betta_tmp += grad;
        gamma_tmp += (val * grad);

        offset += y_stride;
    }

    betta_buffer[threadIdx.x][threadIdx.y] = betta_tmp;
    gamma_buffer[threadIdx.x][threadIdx.y] = gamma_tmp;

    __syncthreads();

    // Sum the shared buffer.
    float s1 = betta_buffer[threadIdx.y][threadIdx.x];
    float s2 = gamma_buffer[threadIdx.y][threadIdx.x];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < TILE_DIM; i <<= 1) {
        s1 += g.shfl_down(s1, i);
        s2 += g.shfl_down(s2, i);
    }

    if (threadIdx.x == 0) {
        betta_grad[pos] = s1;
        gamma_grad[pos] = s2;
    }
}
/*

/* Backward Normalize (Input-Gradient)
 * Using the means and variances from the input
 * This type of backward is invertible!
 * We do the backward using the X_hat (X - u) / sqrt(variance) or the output of Normalization.
 */

template <int row_stride>  // Hidden_Dim
__global__ void LayerNormBackward2(const float* out_grad,
                                   const float* vals_hat,
                                   const float* gamma,
                                   const float* betta,
                                   const float* vars,
                                   float* inp_grad,
                                   bool invertible)
{
    constexpr int iterations = row_stride / THREADS;
    constexpr int iteration_stride = THREADS;  // row_stride / iterations;

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    int row = blockIdx.x;
    int id = threadIdx.x;
    int wid = id / WARP_SIZE;
    constexpr int warp_num = (THREADS < row_stride ? THREADS : row_stride) / WARP_SIZE;
    __shared__ float partialSum[warp_num];

    float vals_arr[iterations];
    float vals_hat_arr[iterations];

#pragma unroll
    for (int i = 0; i < iterations; i++) {
        float gamma_reg = gamma[i * iteration_stride + id];
        vals_arr[i] = out_grad[row * row_stride + i * iteration_stride + id];
        vals_arr[i] *= gamma_reg;
        vals_hat_arr[i] = (invertible ? (vals_hat[row * row_stride + i * iteration_stride + id] -
                                         betta[i * iteration_stride + id]) /
                                            gamma_reg
                                      : vals_hat[row * row_stride + i * iteration_stride + id]);
    }

    float var_reg = vars[row];

    float sum = 0;
    for (int i = 0; i < iterations; i++) {
        sum += vals_hat_arr[i] * vals_arr[i] *
               sqrtf(var_reg);           // dval_hat = gamma * (x - u) * out_grad
        vals_arr[i] *= rsqrtf(var_reg);  // dvar_inv = gamma * out_grad / sqrt(var)
    }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);

    sum = g.shfl(sum, 0);
    sum /= row_stride;

    for (int i = 0; i < iterations; i++) { vals_arr[i] += ((-sum * vals_hat_arr[i]) / var_reg); }

    sum = 0;
    for (int i = 0; i < iterations; i++) { sum += vals_arr[i]; }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);
    sum = g.shfl(sum, 0);
    sum /= row_stride;

    for (int i = 0; i < iterations; i++)
        inp_grad[row * row_stride + i * iteration_stride + id] = (vals_arr[i] - sum);
}

template <int row_stride>  // Hidden_Dim
__global__ void LayerNormBackward2(const __half* out_grad,
                                   const __half* vals_hat,
                                   const __half* gamma,
                                   const __half* betta,
                                   const __half* vars,
                                   __half* inp_grad,
                                   bool invertible)
{
    constexpr int iteration_stride = THREADS / 2;  // row_stride / iterations;
    constexpr int iterations = row_stride / iteration_stride;

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    int row = blockIdx.x;
    int id = threadIdx.x;
    int wid = id / WARP_SIZE;
    constexpr int warp_num =
        (iteration_stride < row_stride ? iteration_stride : row_stride) / WARP_SIZE;
    __shared__ float partialSum[warp_num];

    __half2 vals_arr[iterations];
    float2 vals_arr_f[iterations];
    __half2 vals_hat_arr[iterations];

    __half2* inp_grad_h = reinterpret_cast<__half2*>(inp_grad);
    const __half2* out_grad_h = reinterpret_cast<const __half2*>(out_grad);
    const __half2* vals_hat_h = reinterpret_cast<const __half2*>(vals_hat);

    const __half2* gamma_h = reinterpret_cast<const __half2*>(gamma);
    const __half2* betta_h = (invertible ? reinterpret_cast<const __half2*>(betta) : nullptr);

#pragma unroll
    for (int i = 0; i < iterations; i++) {
        __half2 gamma_reg = gamma_h[i * iteration_stride + id];
        vals_arr[i] = out_grad_h[row * row_stride + i * iteration_stride + id];
        vals_arr[i] *= gamma_reg;
        vals_hat_arr[i] = (invertible ? (vals_hat_h[row * row_stride + i * iteration_stride + id] -
                                         betta_h[i * iteration_stride + id]) /
                                            gamma_reg
                                      : vals_hat_h[row * row_stride + i * iteration_stride + id]);
    }
    __half var_h = vars[row];
    __half2 var_reg = __halves2half2(var_h, var_h);

    float sum = 0.f;
    for (int i = 0; i < iterations; i++) {
        __half2 result_h = (vals_hat_arr[i] * vals_arr[i] * h2sqrt(var_reg));
        float2 result_f = __half22float2(result_h);
        sum += result_f.x;
        sum += result_f.y;
        vals_arr[i] *= h2rsqrt(var_reg);
    }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);

    sum = g.shfl(sum, 0);
    sum /= (2 * row_stride);
    __half2 sum_h = __float2half2_rn(sum);

    for (int i = 0; i < iterations; i++) {
        __half2 temp = ((-sum_h * vals_hat_arr[i]) / (var_reg));
        vals_arr_f[i] = __half22float2(vals_arr[i]);
        float2 temp_f = __half22float2(temp);
        vals_arr_f[i].x += temp_f.x;
        vals_arr_f[i].y += temp_f.y;
    }
    sum = 0.f;

    for (int i = 0; i < iterations; i++) {
        sum += (vals_arr_f[i].x);
        sum += (vals_arr_f[i].y);
    }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);

    sum = g.shfl(sum, 0);
    sum /= (2 * row_stride);

    for (int i = 0; i < iterations; i++) {
        vals_arr_f[i].x -= sum;
        vals_arr_f[i].y -= sum;
        __half2 temp = __float22half2_rn(vals_arr_f[i]);

        inp_grad_h[row * row_stride + i * iteration_stride + id] = temp;
    }
}

template <>
void launch_layerNorm_backward<float>(const float* out_grad,
                                      const float* vals_hat,
                                      const float* vars,
                                      const float* gamma,
                                      float* gamma_grad,
                                      float* betta_grad,
                                      float* inp_grad,
                                      int batch,
                                      int hidden_dim,
                                      hipStream_t stream[2],
                                      bool invertible,
                                      const float* betta)
{
    constexpr int threads = THREADS;

    dim3 grid_dim(hidden_dim / TILE_DIM);
    dim3 block_dim(TILE_DIM, TILE_DIM);

    LayerNormBackward1<float><<<grid_dim, block_dim, 0, stream[0]>>>(
        out_grad, vals_hat, gamma, betta, gamma_grad, betta_grad, batch, hidden_dim, invertible);

    dim3 grid_dim2(batch);
    dim3 block_dim2(threads);

    if (hidden_dim == 768)
        LayerNormBackward2<768><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad, vals_hat, gamma, betta, vars, inp_grad, invertible);
    else if (hidden_dim == 512)
        LayerNormBackward2<512><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad, vals_hat, gamma, betta, vars, inp_grad, invertible);
    else if (hidden_dim == 1024)
        LayerNormBackward2<1024><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad, vals_hat, gamma, betta, vars, inp_grad, invertible);
    else if (hidden_dim == 1536)
        LayerNormBackward2<1536><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad, vals_hat, gamma, betta, vars, inp_grad, invertible);
    else if (hidden_dim == 2048)
        LayerNormBackward2<2048><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad, vals_hat, gamma, betta, vars, inp_grad, invertible);
    else if (hidden_dim == 2560)
        LayerNormBackward2<2560><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad, vals_hat, gamma, betta, vars, inp_grad, invertible);
    else
        throw std::runtime_error("Unsupport hidden_dim.");
}

template <>
void launch_layerNorm_backward<__half>(const __half* out_grad,
                                       const __half* vals_hat,
                                       const __half* vars,
                                       const __half* gamma,
                                       __half* gamma_grad,
                                       __half* betta_grad,
                                       __half* inp_grad,
                                       int batch,
                                       int hidden_dim,
                                       hipStream_t stream[2],
                                       bool invertible,
                                       const __half* betta)
{
    constexpr int threads = THREADS;

    dim3 grid_dim(hidden_dim / TILE_DIM);
    dim3 block_dim(TILE_DIM, TILE_DIM);

    LayerNormBackward1<__half><<<grid_dim, block_dim, 0, stream[0]>>>(
        out_grad, vals_hat, gamma, betta, gamma_grad, betta_grad, batch, hidden_dim, invertible);

    dim3 grid_dim2(batch);
    dim3 block_dim2(threads / 2);

    if (hidden_dim == 768)
        LayerNormBackward2<384><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad, vals_hat, gamma, betta, vars, inp_grad, invertible);
    else if (hidden_dim == 512)
        LayerNormBackward2<256><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad, vals_hat, gamma, betta, vars, inp_grad, invertible);
    else if (hidden_dim == 1024)
        LayerNormBackward2<512><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad, vals_hat, gamma, betta, vars, inp_grad, invertible);
    else if (hidden_dim == 1536)
        LayerNormBackward2<768><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad, vals_hat, gamma, betta, vars, inp_grad, invertible);
    else if (hidden_dim == 2048)
        LayerNormBackward2<1024><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad, vals_hat, gamma, betta, vars, inp_grad, invertible);
    else if (hidden_dim == 2560)
        LayerNormBackward2<1280><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad, vals_hat, gamma, betta, vars, inp_grad, invertible);
    else
        throw std::runtime_error("Unsupport hidden_dim.");
}

/* Backward Normalize (Input-Gradient)
 * Using the means and variances from the input
 * This type of backward is not invertible!
 * We do the backward using the input (X)
 */

template <int row_stride>  // Hidden_Dim
__global__ void LayerNormBackward2(const float* out_grad,
                                   const float* X_vals,
                                   const float* gamma,
                                   const float* vars,
                                   const float* means,
                                   float* inp_grad)
{
    constexpr int iterations = row_stride / THREADS;
    constexpr int iteration_stride = THREADS;  // row_stride / iterations;

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    int row = blockIdx.x;
    int id = threadIdx.x;
    int wid = id / WARP_SIZE;
    constexpr int warp_num = (THREADS < row_stride ? THREADS : row_stride) / WARP_SIZE;
    __shared__ float partialSum[warp_num];

    float vals_arr[iterations];

#pragma unroll
    for (int i = 0; i < iterations; i++) {
        float gamma_reg = gamma[i * iteration_stride + id];
        vals_arr[i] = out_grad[row * row_stride + i * iteration_stride + id];
        vals_arr[i] *= gamma_reg;
    }

    float var_reg = vars[row];
    float mean_reg = means[row];

    float sum = 0;
    float xu[iterations];
    for (int i = 0; i < iterations; i++) {
        xu[i] = (X_vals[row * row_stride + i * iteration_stride + id] - mean_reg);
        sum += vals_arr[i] * xu[i];
        vals_arr[i] *= rsqrtf(var_reg);
    }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);

    sum = g.shfl(sum, 0);
    sum /= row_stride;

    for (int i = 0; i < iterations; i++) {
        vals_arr[i] += (-sum * xu[i] * rsqrtf(var_reg) / (var_reg));
    }

    sum = 0;
    for (int i = 0; i < iterations; i++) { sum += vals_arr[i]; }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);
    sum = g.shfl(sum, 0);
    sum /= row_stride;

    for (int i = 0; i < iterations; i++)
        inp_grad[row * row_stride + i * iteration_stride + id] = (vals_arr[i] - sum);
}

template <int row_stride>  // Hidden_Dim
__global__ void LayerNormBackward2(const __half* out_grad,
                                   const __half* X_vals,
                                   const __half* gamma,
                                   const __half* vars,
                                   const __half* means,
                                   __half* inp_grad)
{
    constexpr int iteration_stride = THREADS / 2;  // row_stride / iterations;
    constexpr int iterations = row_stride / iteration_stride;

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    int row = blockIdx.x;
    int id = threadIdx.x;
    int wid = id / WARP_SIZE;
    constexpr int warp_num =
        (iteration_stride < row_stride ? iteration_stride : row_stride) / WARP_SIZE;

    __shared__ float partialSum[warp_num];

    __half2 vals_arr[iterations];
    float2 vals_arr_f[iterations];

    __half2* inp_grad_h = reinterpret_cast<__half2*>(inp_grad);
    const __half2* out_grad_h = reinterpret_cast<const __half2*>(out_grad);
    const __half2* vals_hat_h = reinterpret_cast<const __half2*>(X_vals);

    const __half2* gamma_h = reinterpret_cast<const __half2*>(gamma);

#pragma unroll
    for (int i = 0; i < iterations; i++) {
        __half2 gamma_reg = gamma_h[i * iteration_stride + id];
        vals_arr[i] = out_grad_h[row * row_stride + i * iteration_stride + id];
        vals_arr[i] *= gamma_reg;  // out_grad * gamma
    }
    __half mean_h = means[row];
    __half var_h = vars[row];
    __half2 var_reg = __halves2half2(var_h, var_h);
    __half2 mean_reg = __halves2half2(mean_h, mean_h);
    __half2 xu[iterations];

    float sum = 0.f;
    for (int i = 0; i < iterations; i++) {
        xu[i] = (vals_hat_h[row * row_stride + i * iteration_stride + id] - mean_reg);
        __half2 result_h = (xu[i] * vals_arr[i]);
        float2 result_f = __half22float2(result_h);
        sum += result_f.x;
        sum += result_f.y;
        vals_arr[i] *= h2rsqrt(var_reg);
    }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);

    sum = g.shfl(sum, 0);
    sum /= (2 * row_stride);
    __half2 sum_h = __float2half2_rn(sum);

    for (int i = 0; i < iterations; i++) {
        __half2 xu_grad = ((-sum_h * xu[i] * h2rsqrt(var_reg)) / (var_reg));
        vals_arr_f[i] = __half22float2(vals_arr[i]);
        float2 xu_grad_f = __half22float2(xu_grad);
        vals_arr_f[i].x += xu_grad_f.x;
        vals_arr_f[i].y += xu_grad_f.y;
    }

    sum = 0.f;
    for (int i = 0; i < iterations; i++) {
        sum += (vals_arr_f[i].x);
        sum += (vals_arr_f[i].y);
    }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);

    sum = g.shfl(sum, 0);
    sum /= (2 * row_stride);

    for (int i = 0; i < iterations; i++) {
        vals_arr_f[i].x -= sum;
        vals_arr_f[i].y -= sum;
        __half2 temp = __float22half2_rn(vals_arr_f[i]);
        inp_grad_h[row * row_stride + i * iteration_stride + id] = temp;
    }
}

template <>
void launch_layerNorm_backward<float>(const float* out_grad,
                                      const float* X_data,
                                      const float* vars,
                                      const float* means,
                                      const float* gamma,
                                      float* gamma_grad,
                                      float* betta_grad,
                                      float* inp_grad,
                                      int batch,
                                      int hidden_dim,
                                      hipStream_t stream[2])
{
    constexpr int threads = THREADS;

    dim3 grid_dim(hidden_dim / TILE_DIM);
    dim3 block_dim(TILE_DIM, TILE_DIM);

    LayerNormBackward1<float><<<grid_dim, block_dim, 0, stream[0]>>>(
        out_grad, X_data, vars, means, gamma_grad, betta_grad, batch, hidden_dim);

    dim3 grid_dim2(batch);
    dim3 block_dim2(threads);

    if (hidden_dim == 768)
        LayerNormBackward2<768><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad, X_data, gamma, vars, means, inp_grad);
    else if (hidden_dim == 512)
        LayerNormBackward2<512><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad, X_data, gamma, vars, means, inp_grad);
    else if (hidden_dim == 1024)
        LayerNormBackward2<1024><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad, X_data, gamma, vars, means, inp_grad);
    else if (hidden_dim == 1536)
        LayerNormBackward2<1536><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad, X_data, gamma, vars, means, inp_grad);
    else if (hidden_dim == 2048)
        LayerNormBackward2<2048><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad, X_data, gamma, vars, means, inp_grad);
    else if (hidden_dim == 2560)
        LayerNormBackward2<2560><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad, X_data, gamma, vars, means, inp_grad);
    else
        throw std::runtime_error("Unsupport hidden_dim.");
}

template <>
void launch_layerNorm_backward<__half>(const __half* out_grad,
                                       const __half* X_data,
                                       const __half* vars,
                                       const __half* means,
                                       const __half* gamma,
                                       __half* gamma_grad,
                                       __half* betta_grad,
                                       __half* inp_grad,
                                       int batch,
                                       int hidden_dim,
                                       hipStream_t stream[2])
{
    constexpr int threads = THREADS;

    dim3 grid_dim(hidden_dim / TILE_DIM);
    dim3 block_dim(TILE_DIM, TILE_DIM);

    LayerNormBackward1<__half><<<grid_dim, block_dim, 0, stream[0]>>>(
        out_grad, X_data, vars, means, gamma_grad, betta_grad, batch, hidden_dim);

    dim3 grid_dim2(batch);
    dim3 block_dim2(threads / 2);

    if (hidden_dim == 768)
        LayerNormBackward2<384><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad, X_data, gamma, vars, means, inp_grad);
    else if (hidden_dim == 512)
        LayerNormBackward2<256><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad, X_data, gamma, vars, means, inp_grad);
    else if (hidden_dim == 1024)
        LayerNormBackward2<512><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad, X_data, gamma, vars, means, inp_grad);
    else if (hidden_dim == 1536)
        LayerNormBackward2<768><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad, X_data, gamma, vars, means, inp_grad);
    else if (hidden_dim == 2048)
        LayerNormBackward2<1024><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad, X_data, gamma, vars, means, inp_grad);
    else if (hidden_dim == 2560)
        LayerNormBackward2<1280><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad, X_data, gamma, vars, means, inp_grad);
    else
        throw std::runtime_error("Unsupport hidden_dim.");
}

template <typename T>
__global__ void LayerNormBackward1_fused_add(const T* __restrict__ out_grad1,
                                             const T* __restrict__ out_grad2,
                                             const T* __restrict__ vals_hat,
                                             const T* __restrict__ gamma,
                                             const T* __restrict__ betta,
                                             T* __restrict__ gamma_grad,
                                             T* __restrict__ betta_grad,
                                             int rows,
                                             int width,
                                             bool invertible)
{
    __shared__ float betta_buffer[TILE_DIM][TILE_DIM + 1];
    __shared__ float gamma_buffer[TILE_DIM][TILE_DIM + 1];

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<TILE_DIM> g = cg::tiled_partition<TILE_DIM>(b);

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = threadIdx.y * width + idx;
    int y_stride = width * TILE_DIM;

    int pos = blockIdx.x * TILE_DIM + threadIdx.y;
    float betta_reg = (invertible ? (float)betta[pos] : 0.0f);
    float gamma_reg = (float)gamma[pos];

    // Loop across matrix height
    float betta_tmp = 0;
    float gamma_tmp = 0;
    for (int r = threadIdx.y; r < rows; r += TILE_DIM) {
        float grad = (float)out_grad1[offset] + (float)out_grad2[offset];
        float val = (invertible ? ((float)vals_hat[offset] - betta_reg) / gamma_reg
                                : (float)vals_hat[offset]);
        betta_tmp += grad;
        gamma_tmp += (val * grad);

        offset += y_stride;
    }

    betta_buffer[threadIdx.x][threadIdx.y] = betta_tmp;
    gamma_buffer[threadIdx.x][threadIdx.y] = gamma_tmp;

    __syncthreads();

    // Sum the shared buffer.
    float s1 = betta_buffer[threadIdx.y][threadIdx.x];
    float s2 = gamma_buffer[threadIdx.y][threadIdx.x];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < TILE_DIM; i <<= 1) {
        s1 += g.shfl_down(s1, i);
        s2 += g.shfl_down(s2, i);
    }

    if (threadIdx.x == 0) {
        betta_grad[pos] = s1;
        gamma_grad[pos] = s2;
    }
}

template <typename T>
__global__ void LayerNormBackward1_fused_add(const T* __restrict__ out_grad1,
                                             const T* __restrict__ out_grad2,
                                             const T* __restrict__ X_data,
                                             const T* __restrict__ vars,
                                             const T* __restrict__ means,
                                             T* __restrict__ gamma_grad,
                                             T* __restrict__ betta_grad,
                                             int rows,
                                             int width)
{
    __shared__ float betta_buffer[TILE_DIM][TILE_DIM + 1];
    __shared__ float gamma_buffer[TILE_DIM][TILE_DIM + 1];

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<TILE_DIM> g = cg::tiled_partition<TILE_DIM>(b);

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = threadIdx.y * width + idx;
    int y_stride = width * TILE_DIM;

    int pos = blockIdx.x * TILE_DIM + threadIdx.y;
    // Loop across matrix height

    float betta_tmp = 0;
    float gamma_tmp = 0;
    for (int r = threadIdx.y; r < rows; r += TILE_DIM) {
        float grad = (float)out_grad1[offset] + (float)out_grad2[offset];
        float val = (float)X_data[offset];
        val = (val - (float)means[r]) * rsqrtf((float)vars[r]);
        betta_tmp += grad;
        gamma_tmp += (val * grad);

        offset += y_stride;
    }

    betta_buffer[threadIdx.x][threadIdx.y] = betta_tmp;
    gamma_buffer[threadIdx.x][threadIdx.y] = gamma_tmp;

    __syncthreads();

    // Sum the shared buffer.
    float s1 = betta_buffer[threadIdx.y][threadIdx.x];
    float s2 = gamma_buffer[threadIdx.y][threadIdx.x];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < TILE_DIM; i <<= 1) {
        s1 += g.shfl_down(s1, i);
        s2 += g.shfl_down(s2, i);
    }

    if (threadIdx.x == 0) {
        betta_grad[pos] = s1;
        gamma_grad[pos] = s2;
    }
}

template <int row_stride>  // Hidden_Dim
__global__ void LayerNormBackward2_fused_add(const float* out_grad1,
                                             const float* out_grad2,
                                             const float* vals_hat,
                                             const float* gamma,
                                             const float* betta,
                                             const float* vars,
                                             float* inp_grad,
                                             bool invertible)
{
    constexpr int iterations = row_stride / THREADS;
    constexpr int iteration_stride = THREADS;  // row_stride / iterations;

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    int row = blockIdx.x;
    int id = threadIdx.x;
    int wid = id / WARP_SIZE;
    constexpr int warp_num = (THREADS < row_stride ? THREADS : row_stride) / WARP_SIZE;
    __shared__ float partialSum[warp_num];

    float vals_arr[iterations];
    float vals_hat_arr[iterations];

#pragma unroll
    for (int i = 0; i < iterations; i++) {
        float gamma_reg = gamma[i * iteration_stride + id];
        vals_arr[i] = out_grad1[row * row_stride + i * iteration_stride + id];
        vals_arr[i] *= gamma_reg;
        vals_hat_arr[i] = (invertible ? (vals_hat[row * row_stride + i * iteration_stride + id] -
                                         betta[i * iteration_stride + id]) /
                                            gamma_reg
                                      : vals_hat[row * row_stride + i * iteration_stride + id]);
    }

    float var_reg = vars[row];

    float sum = 0;
    for (int i = 0; i < iterations; i++) {
        sum += vals_hat_arr[i] * vals_arr[i] * sqrtf(var_reg);
        vals_arr[i] *= rsqrtf(var_reg);
    }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);

    sum = g.shfl(sum, 0);
    sum /= row_stride;

    for (int i = 0; i < iterations; i++) { vals_arr[i] += ((-sum * vals_hat_arr[i]) / var_reg); }

    sum = 0;
    for (int i = 0; i < iterations; i++) { sum += vals_arr[i]; }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);
    sum = g.shfl(sum, 0);
    sum /= row_stride;

    for (int i = 0; i < iterations; i++)
        inp_grad[row * row_stride + i * iteration_stride + id] =
            (vals_arr[i] - sum) + out_grad2[row * row_stride + i * iteration_stride + id];
}

template <int row_stride>  // Hidden_Dim
__global__ void LayerNormBackward2_fused_add(const __half* out_grad1,
                                             const __half* out_grad2,
                                             const __half* vals_hat,
                                             const __half* gamma,
                                             const __half* betta,
                                             const __half* vars,
                                             __half* inp_grad,
                                             bool invertible)
{
    constexpr int iteration_stride = THREADS / 2;  // row_stride / iterations;
    constexpr int iterations = row_stride / iteration_stride;

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    int row = blockIdx.x;
    int id = threadIdx.x;
    int wid = id / WARP_SIZE;
    constexpr int warp_num =
        (iteration_stride < row_stride ? iteration_stride : row_stride) / WARP_SIZE;
    __shared__ float partialSum[warp_num];

    __half2 vals_arr[iterations];
    float2 vals_arr_f[iterations];
    __half2 vals_hat_arr[iterations];

    // float2 result[iterations];

    __half2* inp_grad_h = reinterpret_cast<__half2*>(inp_grad);
    const __half2* out_grad_h1 = reinterpret_cast<const __half2*>(out_grad1);
    const __half2* out_grad_h2 = reinterpret_cast<const __half2*>(out_grad2);
    const __half2* vals_hat_h = reinterpret_cast<const __half2*>(vals_hat);

    const __half2* gamma_h = reinterpret_cast<const __half2*>(gamma);
    const __half2* betta_h = (invertible ? reinterpret_cast<const __half2*>(betta) : nullptr);

#pragma unroll
    for (int i = 0; i < iterations; i++) {
        __half2 gamma_reg = gamma_h[i * iteration_stride + id];
        vals_arr[i] = out_grad_h1[row * row_stride + i * iteration_stride + id];
        vals_arr[i] *= gamma_reg;  // out_grad * gamma
        vals_hat_arr[i] = (invertible ? (vals_hat_h[row * row_stride + i * iteration_stride + id] -
                                         betta_h[i * iteration_stride + id]) /
                                            gamma_reg
                                      : vals_hat_h[row * row_stride + i * iteration_stride + id]);
    }
    __half var_h = vars[row];
    __half2 var_reg = __halves2half2(var_h, var_h);

    float sum = 0.f;
    for (int i = 0; i < iterations; i++) {
        __half2 result_h = (vals_hat_arr[i] * vals_arr[i] * h2sqrt(var_reg));
        float2 result_f = __half22float2(result_h);
        sum += result_f.x;
        sum += result_f.y;
        vals_arr[i] *= h2rsqrt(var_reg);
    }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);

    sum = g.shfl(sum, 0);
    sum /= (2 * row_stride);
    __half2 sum_h = __float2half2_rn(sum);

    for (int i = 0; i < iterations; i++) {
        __half2 temp = ((-sum_h * vals_hat_arr[i]) / (var_reg));
        vals_arr_f[i] = __half22float2(vals_arr[i]);
        float2 temp_f = __half22float2(temp);
        vals_arr_f[i].x += temp_f.x;
        vals_arr_f[i].y += temp_f.y;
    }
    sum = 0.f;
    for (int i = 0; i < iterations; i++) {
        sum += (vals_arr_f[i].x);
        sum += (vals_arr_f[i].y);
    }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);

    sum = g.shfl(sum, 0);
    sum /= (2 * row_stride);

    for (int i = 0; i < iterations; i++) {
        vals_arr_f[i].x -= sum;
        vals_arr_f[i].y -= sum;
        __half2 temp = __float22half2_rn(vals_arr_f[i]);

        inp_grad_h[row * row_stride + i * iteration_stride + id] =
            temp + out_grad_h2[row * row_stride + i * iteration_stride + id];
    }
}

template <>
void launch_layerNorm_backward_fused_add<float>(const float* out_grad1,
                                                const float* out_grad2,
                                                const float* vals_hat,
                                                const float* vars,
                                                const float* gamma,
                                                float* gamma_grad,
                                                float* betta_grad,
                                                float* inp_grad,
                                                int batch,
                                                int hidden_dim,
                                                hipStream_t stream[2],
                                                bool invertible,
                                                const float* betta)
{
    constexpr int threads = THREADS;

    dim3 grid_dim(hidden_dim / TILE_DIM);
    dim3 block_dim(TILE_DIM, TILE_DIM);
    LayerNormBackward1<float><<<grid_dim, block_dim, 0, stream[0]>>>(
        out_grad1, vals_hat, gamma, betta, gamma_grad, betta_grad, batch, hidden_dim, invertible);

    dim3 grid_dim2(batch);
    dim3 block_dim2(threads);

    if (hidden_dim == 768)
        LayerNormBackward2_fused_add<768><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad1, out_grad2, vals_hat, gamma, betta, vars, inp_grad, invertible);
    else if (hidden_dim == 512)
        LayerNormBackward2_fused_add<512><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad1, out_grad2, vals_hat, gamma, betta, vars, inp_grad, invertible);
    else if (hidden_dim == 1024)
        LayerNormBackward2_fused_add<1024><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad1, out_grad2, vals_hat, gamma, betta, vars, inp_grad, invertible);
    else if (hidden_dim == 1536)
        LayerNormBackward2_fused_add<1536><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad1, out_grad2, vals_hat, gamma, betta, vars, inp_grad, invertible);
    else if (hidden_dim == 2048)
        LayerNormBackward2_fused_add<2048><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad1, out_grad2, vals_hat, gamma, betta, vars, inp_grad, invertible);
    else if (hidden_dim == 2560)
        LayerNormBackward2_fused_add<2560><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad1, out_grad2, vals_hat, gamma, betta, vars, inp_grad, invertible);
    else
        throw std::runtime_error("Unsupport hidden_dim.");
}

template <>
void launch_layerNorm_backward_fused_add<__half>(const __half* out_grad1,
                                                 const __half* out_grad2,
                                                 const __half* vals_hat,
                                                 const __half* vars,
                                                 const __half* gamma,
                                                 __half* gamma_grad,
                                                 __half* betta_grad,
                                                 __half* inp_grad,
                                                 int batch,
                                                 int hidden_dim,
                                                 hipStream_t stream[2],
                                                 bool invertible,
                                                 const __half* betta)
{
    constexpr int threads = THREADS;

    dim3 grid_dim(hidden_dim / TILE_DIM);
    dim3 block_dim(TILE_DIM, TILE_DIM);

    LayerNormBackward1<__half><<<grid_dim, block_dim, 0, stream[0]>>>(
        out_grad1, vals_hat, gamma, betta, gamma_grad, betta_grad, batch, hidden_dim, invertible);

    dim3 grid_dim2(batch);
    dim3 block_dim2(threads / 2);

    if (hidden_dim == 768)
        LayerNormBackward2_fused_add<384><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad1, out_grad2, vals_hat, gamma, betta, vars, inp_grad, invertible);
    else if (hidden_dim == 512)
        LayerNormBackward2_fused_add<256><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad1, out_grad2, vals_hat, gamma, betta, vars, inp_grad, invertible);
    else if (hidden_dim == 1024)
        LayerNormBackward2_fused_add<512><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad1, out_grad2, vals_hat, gamma, betta, vars, inp_grad, invertible);
    else if (hidden_dim == 1536)
        LayerNormBackward2_fused_add<768><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad1, out_grad2, vals_hat, gamma, betta, vars, inp_grad, invertible);
    else if (hidden_dim == 2048)
        LayerNormBackward2_fused_add<1024><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad1, out_grad2, vals_hat, gamma, betta, vars, inp_grad, invertible);
    else if (hidden_dim == 2560)
        LayerNormBackward2_fused_add<1280><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad1, out_grad2, vals_hat, gamma, betta, vars, inp_grad, invertible);
    else
        throw std::runtime_error("Unsupport hidden_dim.");
}

/* Backward Normalize (Input-Gradient)
 * Using the means and variances from the input
 * This type of backward is not invertible!
 * We do the backward using the input (X)
 */

template <int row_stride>  // Hidden_Dim
__global__ void LayerNormBackward2_fused_add(const float* out_grad1,
                                             const float* out_grad2,
                                             const float* X_vals,
                                             const float* gamma,
                                             const float* vars,
                                             const float* means,
                                             float* inp_grad)
{
    constexpr int iterations = row_stride / THREADS;
    constexpr int iteration_stride = THREADS;  // row_stride / iterations;

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    int row = blockIdx.x;
    int id = threadIdx.x;
    int wid = id / WARP_SIZE;
    constexpr int warp_num = (THREADS < row_stride ? THREADS : row_stride) / WARP_SIZE;
    __shared__ float partialSum[warp_num];

    float vals_arr[iterations];
    float vals_hat_arr[iterations];

#pragma unroll
    for (int i = 0; i < iterations; i++) {
        float gamma_reg = gamma[i * iteration_stride + id];
        vals_arr[i] = out_grad1[row * row_stride + i * iteration_stride + id];
        vals_arr[i] *= gamma_reg;
        vals_hat_arr[i] = X_vals[row * row_stride + i * iteration_stride + id];
    }

    float var_reg = vars[row];
    float mean_reg = means[row];

    float sum = 0;
    float xu[iterations];
    for (int i = 0; i < iterations; i++) {
        xu[i] = (vals_hat_arr[i] - mean_reg);
        sum += vals_arr[i] * xu[i];
        vals_arr[i] *= rsqrtf(var_reg);
    }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);

    sum = g.shfl(sum, 0);
    sum /= row_stride;

    for (int i = 0; i < iterations; i++) {
        vals_arr[i] += (-sum * xu[i] * rsqrtf(var_reg) / (var_reg));
    }

    sum = 0;
    for (int i = 0; i < iterations; i++) { sum += vals_arr[i]; }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);
    sum = g.shfl(sum, 0);
    sum /= row_stride;

    for (int i = 0; i < iterations; i++)
        inp_grad[row * row_stride + i * iteration_stride + id] =
            (vals_arr[i] - sum) + out_grad2[row * row_stride + i * iteration_stride + id];
    ;
}

template <int row_stride>  // Hidden_Dim
__global__ void LayerNormBackward2_fused_add(const __half* out_grad1,
                                             const __half* out_grad2,
                                             const __half* X_vals,
                                             const __half* gamma,
                                             const __half* vars,
                                             const __half* means,
                                             __half* inp_grad)
{
    constexpr int iteration_stride = THREADS / 2;  // row_stride / iterations;
    constexpr int iterations = row_stride / iteration_stride;

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    int row = blockIdx.x;
    int id = threadIdx.x;
    int wid = id / WARP_SIZE;
    constexpr int warp_num =
        (iteration_stride < row_stride ? iteration_stride : row_stride) / WARP_SIZE;

    __shared__ float partialSum[warp_num];

    __half2 vals_arr[iterations];
    float2 vals_arr_f[iterations];
    __half2 vals_hat_arr[iterations];

    __half2* inp_grad_h = reinterpret_cast<__half2*>(inp_grad);
    const __half2* out_grad_h1 = reinterpret_cast<const __half2*>(out_grad1);
    const __half2* out_grad_h2 = reinterpret_cast<const __half2*>(out_grad2);
    const __half2* vals_hat_h = reinterpret_cast<const __half2*>(X_vals);

    const __half2* gamma_h = reinterpret_cast<const __half2*>(gamma);

#pragma unroll
    for (int i = 0; i < iterations; i++) {
        __half2 gamma_reg = gamma_h[i * iteration_stride + id];
        vals_arr[i] = out_grad_h1[row * row_stride + i * iteration_stride + id];
        vals_arr[i] *= gamma_reg;  // out_grad * gamma
        vals_hat_arr[i] = vals_hat_h[row * row_stride + i * iteration_stride + id];
    }

    __half mean_h = means[row];
    __half var_h = vars[row];
    __half2 var_reg = __halves2half2(var_h, var_h);
    __half2 mean_reg = __halves2half2(mean_h, mean_h);
    __half2 xu[iterations];

    float sum = 0.f;
    for (int i = 0; i < iterations; i++) {
        xu[i] = (vals_hat_arr[i] - mean_reg);
        __half2 result_h = (xu[i] * vals_arr[i]);
        float2 result_f = __half22float2(result_h);
        sum += result_f.x;
        sum += result_f.y;
        vals_arr[i] *= h2rsqrt(var_reg);
    }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);

    sum = g.shfl(sum, 0);
    sum /= (2 * row_stride);
    __half2 sum_h = __float2half2_rn(sum);

    for (int i = 0; i < iterations; i++) {
        __half2 xu_grad = ((-sum_h * xu[i] * h2rsqrt(var_reg)) / (var_reg));
        vals_arr_f[i] = __half22float2(vals_arr[i]);
        float2 xu_grad_f = __half22float2(xu_grad);
        vals_arr_f[i].x += xu_grad_f.x;
        vals_arr_f[i].y += xu_grad_f.y;
    }

    sum = 0.f;
    for (int i = 0; i < iterations; i++) {
        sum += (vals_arr_f[i].x);
        sum += (vals_arr_f[i].y);
    }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);

    sum = g.shfl(sum, 0);
    sum /= (2 * row_stride);

    for (int i = 0; i < iterations; i++) {
        vals_arr_f[i].x -= sum;
        vals_arr_f[i].y -= sum;
        __half2 temp = __float22half2_rn(vals_arr_f[i]);
        inp_grad_h[row * row_stride + i * iteration_stride + id] =
            temp + out_grad_h2[row * row_stride + i * iteration_stride + id];
    }
}

template <>
void launch_layerNorm_backward_fused_add<float>(const float* out_grad1,
                                                const float* out_grad2,
                                                const float* X_data,
                                                const float* vars,
                                                const float* means,
                                                const float* gamma,
                                                float* gamma_grad,
                                                float* betta_grad,
                                                float* inp_grad,
                                                int batch,
                                                int hidden_dim,
                                                hipStream_t stream[2])
{
    constexpr int threads = THREADS;

    dim3 grid_dim(hidden_dim / TILE_DIM);
    dim3 block_dim(TILE_DIM, TILE_DIM);

    LayerNormBackward1<float><<<grid_dim, block_dim, 0, stream[0]>>>(
        out_grad1, X_data, vars, means, gamma_grad, betta_grad, batch, hidden_dim);

    dim3 grid_dim2(batch);
    dim3 block_dim2(threads);

    if (hidden_dim == 768)
        LayerNormBackward2_fused_add<768><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad1, out_grad2, X_data, gamma, vars, means, inp_grad);
    else if (hidden_dim == 512)
        LayerNormBackward2_fused_add<512><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad1, out_grad2, X_data, gamma, vars, means, inp_grad);
    else if (hidden_dim == 1024)
        LayerNormBackward2_fused_add<1024><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad1, out_grad2, X_data, gamma, vars, means, inp_grad);
    else if (hidden_dim == 1536)
        LayerNormBackward2_fused_add<1536><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad1, out_grad2, X_data, gamma, vars, means, inp_grad);
    else if (hidden_dim == 2048)
        LayerNormBackward2_fused_add<2048><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad1, out_grad2, X_data, gamma, vars, means, inp_grad);
    else if (hidden_dim == 2560)
        LayerNormBackward2_fused_add<2560><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad1, out_grad2, X_data, gamma, vars, means, inp_grad);
    else
        throw std::runtime_error("Unsupport hidden_dim.");
}

template <>
void launch_layerNorm_backward_fused_add<__half>(const __half* out_grad1,
                                                 const __half* out_grad2,
                                                 const __half* X_data,
                                                 const __half* vars,
                                                 const __half* means,
                                                 const __half* gamma,
                                                 __half* gamma_grad,
                                                 __half* betta_grad,
                                                 __half* inp_grad,
                                                 int batch,
                                                 int hidden_dim,
                                                 hipStream_t stream[2])
{
    constexpr int threads = THREADS;

    dim3 grid_dim(hidden_dim / TILE_DIM);
    dim3 block_dim(TILE_DIM, TILE_DIM);

    LayerNormBackward1<__half><<<grid_dim, block_dim, 0, stream[0]>>>(
        out_grad1, X_data, vars, means, gamma_grad, betta_grad, batch, hidden_dim);

    dim3 grid_dim2(batch);
    dim3 block_dim2(threads / 2);

    if (hidden_dim == 768)
        LayerNormBackward2_fused_add<384><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad1, out_grad2, X_data, gamma, vars, means, inp_grad);
    else if (hidden_dim == 512)
        LayerNormBackward2_fused_add<256><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad1, out_grad2, X_data, gamma, vars, means, inp_grad);
    else if (hidden_dim == 1024)
        LayerNormBackward2_fused_add<512><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad1, out_grad2, X_data, gamma, vars, means, inp_grad);
    else if (hidden_dim == 1536)
        LayerNormBackward2_fused_add<768><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad1, out_grad2, X_data, gamma, vars, means, inp_grad);
    else if (hidden_dim == 2048)
        LayerNormBackward2_fused_add<1024><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad1, out_grad2, X_data, gamma, vars, means, inp_grad);
    else if (hidden_dim == 2560)
        LayerNormBackward2_fused_add<1280><<<grid_dim2, block_dim2, 0, stream[1]>>>(
            out_grad1, out_grad2, X_data, gamma, vars, means, inp_grad);
    else
        throw std::runtime_error("Unsupport hidden_dim.");
}

#include "hip/hip_runtime.h"
#include <unistd.h>

#include <hip/hip_runtime.h>
#include "hipblas.h"

#include "buffalo/cuda/utils.cuh"
#include "buffalo/cuda/als/als.hpp"


namespace cuda_als{

using std::invalid_argument;
using namespace cuda_buffalo;

__global__ void least_squares_cg_kernel(const int dim, const int vdim, 
        const int rows, const int op_rows, 
        float* P, const float* Q, const float* FF, float* loss_nume, float* loss_deno,
        const int start_x, const int next_x,
        const int64_t* indptr, const int* keys, const float* vals, 
        const float alpha, const float reg, const bool adaptive_reg, const float cg_tolerance,
        const int num_cg_max_iters, const bool compute_loss,
        const float eps, const bool axis){
    extern __shared__ float shared_memory[];
    float* Ap = &shared_memory[0];
    float* r = &shared_memory[vdim];
    float* p = &shared_memory[2*vdim];
    
    int64_t shift = start_x == 0? 0: indptr[start_x - 1];
    for (int row=blockIdx.x + start_x; row<next_x; row+=gridDim.x){
        float* _P = &P[row*vdim];
        
        // assume that shifted index can be represented by size_t
        size_t beg = row == 0? 0: indptr[row - 1] - shift;
        size_t end = indptr[row] - shift;

        if (beg == end) {
            _P[threadIdx.x] = 0;
            continue;
        }
        // set adaptive regularization coefficient
        float ada_reg = adaptive_reg? (end - beg): 1.0;
        ada_reg *= reg;

        float tmp = 0.0;
        // not necessary to compute vdim times
        for (int d=0; d<dim; ++d)
            tmp -= _P[d] * FF[d * vdim + threadIdx.x];
        Ap[threadIdx.x] = -tmp;

        // compute loss on negative samples (only item side)
        if (compute_loss and axis){
            float _dot = dot(_P, Ap);
            if (threadIdx.x == 0){
                loss_nume[blockIdx.x] += _dot;
                loss_deno[blockIdx.x] += op_rows;
            }
        }

        tmp -= _P[threadIdx.x] * ada_reg;

        for (size_t idx=beg; idx<end; ++idx){
            const float* _Q = &Q[keys[idx] * vdim];
            const float v = vals[idx];
            float _dot = dot(_P, _Q);
            // compute loss on positive samples (only item side)
            if (compute_loss and axis and threadIdx.x == 0){
                loss_nume[blockIdx.x] -= _dot * _dot;
                loss_nume[blockIdx.x] += (1.0 + v * alpha) * (_dot - 1) * (_dot - 1);
                loss_deno[blockIdx.x] += v * alpha;
            }
            tmp += (1 + alpha * v * (1 - _dot)) * _Q[threadIdx.x];
        }
        p[threadIdx.x] = r[threadIdx.x] = tmp;

        float rsold = dot(r, r);
        // early stopping
        if (rsold < cg_tolerance){
            // compute loss on regularization (both user and item side)
            if (compute_loss){
                float _dot = dot(_P, _P);
                if (threadIdx.x == 0)
                    loss_nume[blockIdx.x] += _dot * ada_reg;
            }
            continue;
        }

        // iterate cg
        for (int it=0; it<num_cg_max_iters; ++it){
            Ap[threadIdx.x] = ada_reg * p[threadIdx.x];
            for (int d=0; d<dim; ++d){
                Ap[threadIdx.x] += p[d] * FF[d * vdim + threadIdx.x];
            }
            for (size_t idx=beg; idx<end; ++idx){
                const float* _Q = &Q[keys[idx] * vdim];
                const float v = vals[idx];
                float _dot = dot(p, _Q);
                Ap[threadIdx.x] += v * alpha * _dot * _Q[threadIdx.x];
            }
            float alpha = rsold / (dot(p, Ap) + eps);
            _P[threadIdx.x] += alpha * p[threadIdx.x];
            r[threadIdx.x] -= alpha * Ap[threadIdx.x];
            float rsnew = dot(r, r);
            if (rsnew < cg_tolerance) break;
            p[threadIdx.x] = r[threadIdx.x] + (rsnew / (rsold + eps)) * p[threadIdx.x];
            rsold = rsnew;
            __syncthreads();
        }

        // compute loss on regularization (both user and item side)
        if (compute_loss){
            float _dot = dot(_P, _P);
            if (threadIdx.x == 0)
                loss_nume[blockIdx.x] += _dot * ada_reg;
        }
        
        if (isnan(rsold)){
            if (threadIdx.x == 0)
                printf("Warning NaN detected in row %d of %d\n", row, rows);
            _P[threadIdx.x] = 0.0;
        }
    }
}

CuALS::CuALS(){
    logger_ = BuffaloLogger().get_logger();
    opt_setted_ = false, initialized_ = false, ph_setted_ = false;
    
    CHECK_CUDA(hipGetDevice(&devId_));
    hipDeviceProp_t prop;
    CHECK_CUDA(hipGetDeviceProperties(&prop, devId_));
    mp_cnt_ = prop.multiProcessorCount;
    int major = prop.major;
    int minor = prop.minor;
    cores_ = -1;

    switch (major){
        case 2: // Fermi
            if (minor == 1) cores_ = mp_cnt_ * 48;
            else cores_ = mp_cnt_ * 32;
            break;
        case 3: // Kepler
            cores_ = mp_cnt_ * 192;
            break;
        case 5: // Maxwell
            cores_ = mp_cnt_ * 128;
            break;
        case 6: // Pascal
            if (minor == 1) cores_ = mp_cnt_ * 128;
            else if (minor == 0) cores_ = mp_cnt_ * 64;
            else INFO0("Unknown device type");
            break;
        case 7: // Volta
            if (minor == 0) cores_ = mp_cnt_ * 64;
            else INFO0("Unknown device type");
            break;
        default:
            INFO0("Unknown device type"); 
            break;
    }

    if (cores_ == -1) cores_ = mp_cnt_ * 128;
    INFO("cuda device info, major: {}, minor: {}, multi processors: {}, cores: {}",
         major, minor, mp_cnt_, cores_);
}

CuALS::~CuALS(){
    // destructor
    CHECK_CUBLAS(hipblasDestroy(blas_handle_));

    _release_utility();
    _release_embedding();
    _release_placeholder();

}

void CuALS::_release_utility(){
    // free memory of utility variables
    if (opt_setted_){
        CHECK_CUDA(hipFree(devFF_)); devFF_ = nullptr;
        if (compute_loss_){
            free(hostLossNume_);
            free(hostLossDeno_);
            CHECK_CUDA(hipFree(devLossNume_));
            CHECK_CUDA(hipFree(devLossDeno_));
        }
    }

    opt_setted_ = false;
}

void CuALS::_release_embedding(){
    // free memory of embedding matrix
    if (initialized_){
        CHECK_CUDA(hipFree(devP_));
        CHECK_CUDA(hipFree(devQ_));
        devP_ = nullptr, devQ_ = nullptr;
        hostP_ = nullptr, hostQ_ = nullptr;
    }
    initialized_ = false;
}

void CuALS::_release_placeholder(){
    // free memory of placeholders
    if (ph_setted_){
        CHECK_CUDA(hipFree(lindptr_));
        CHECK_CUDA(hipFree(rindptr_));
        CHECK_CUDA(hipFree(keys_));
        CHECK_CUDA(hipFree(vals_));
    }
    ph_setted_ = false;
}

bool CuALS::parse_option(std::string opt_path, Json& j){
    std::ifstream in(opt_path.c_str());
    if (not in.is_open()) {
        return false;
    }

    std::string str((std::istreambuf_iterator<char>(in)),
               std::istreambuf_iterator<char>());
    std::string err_cmt;
    auto _j = Json::parse(str, err_cmt);
    if (not err_cmt.empty()) {
        return false;
    }
    j = _j;
    return true;
}

bool CuALS::init(std::string opt_path){
    // parse options
    bool ok = parse_option(opt_path, opt_);
    if (ok){
        // if already setted, free memory
        _release_utility();

        // set options
        compute_loss_ = opt_["compute_loss_on_training"].bool_value();
        adaptive_reg_ = opt_["adaptive_reg"].bool_value();

        dim_ = opt_["d"].int_value();
        num_cg_max_iters_ = opt_["num_cg_max_iters"].int_value();
         
        alpha_ = opt_["alpha"].number_value();
        reg_u_ = opt_["reg_u"].number_value();
        reg_i_ = opt_["reg_i"].number_value();
        cg_tolerance_ = opt_["cg_tolerance"].number_value();
        eps_ = opt_["eps"].number_value();
       
        // virtual dimension
        vdim_ = (dim_ / WARP_SIZE) * WARP_SIZE;
        if (dim_ % WARP_SIZE > 0) vdim_ += WARP_SIZE;
        CHECK_CUDA(hipMalloc(&devFF_, sizeof(float)*vdim_*vdim_));
        CHECK_CUBLAS(hipblasCreate(&blas_handle_));
       
        block_cnt_ = opt_["hyper_threads"].int_value() * (cores_ / vdim_);

        if (compute_loss_){
            hostLossNume_ = (float*) malloc(sizeof(float)*block_cnt_);
            hostLossDeno_ = (float*) malloc(sizeof(float)*block_cnt_);
            CHECK_CUDA(hipMalloc(&devLossNume_, sizeof(float)*block_cnt_));
            CHECK_CUDA(hipMalloc(&devLossDeno_, sizeof(float)*block_cnt_));
        }
        opt_setted_ = true;
    }
    return ok;
}

void CuALS::initialize_model(
        float* P, int P_rows,
        float* Q, int Q_rows)
{    
    // if already setted, free memory
    _release_embedding();
    
    // initialize parameters and send to gpu memory
    hostP_ = P;
    hostQ_ = Q;
    P_rows_ = P_rows;
    Q_rows_ = Q_rows;
    CHECK_CUDA(hipMalloc(&devP_, sizeof(float)*P_rows_*vdim_));
    CHECK_CUDA(hipMemcpy(devP_, hostP_, sizeof(float)*P_rows_*vdim_, 
               hipMemcpyHostToDevice));
    CHECK_CUDA(hipMalloc(&devQ_, sizeof(float)*Q_rows_*vdim_));
    CHECK_CUDA(hipMemcpy(devQ_, hostQ_, sizeof(float)*Q_rows_*vdim_, 
               hipMemcpyHostToDevice));
    CHECK_CUDA(hipDeviceSynchronize());
    initialized_ = true;
}

void CuALS::set_placeholder(int64_t* lindptr, int64_t* rindptr, size_t batch_size)
{
    // if already setted, free memory
    _release_placeholder();
    
    CHECK_CUDA(hipMalloc(&lindptr_, sizeof(int64_t)*(P_rows_)));
    CHECK_CUDA(hipMalloc(&rindptr_, sizeof(int64_t)*(Q_rows_)));
    CHECK_CUDA(hipMemcpy(lindptr_, lindptr, sizeof(int64_t)*(P_rows_), 
            hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(rindptr_, rindptr, sizeof(int64_t)*(Q_rows_), 
            hipMemcpyHostToDevice));

    CHECK_CUDA(hipMalloc(&keys_, sizeof(int)*batch_size));
    CHECK_CUDA(hipMalloc(&vals_, sizeof(float)*batch_size));
    batch_size_ = batch_size;
    ph_setted_ = true;
}


void CuALS::precompute(int axis){
    // precompute FF using cublas
    int op_rows = axis == 0? Q_rows_: P_rows_;
    float* opF = axis == 0? devQ_: devP_;
    float alpha = 1.0, beta = 0.0;
    CHECK_CUBLAS(hipblasSgemm(blas_handle_, HIPBLAS_OP_N, HIPBLAS_OP_T,
                 vdim_, vdim_, op_rows, &alpha, 
                 opF, vdim_, opF, vdim_, &beta, devFF_, vdim_));
    CHECK_CUDA(hipDeviceSynchronize());
}

void CuALS::_synchronize(int start_x, int next_x, int axis, bool device_to_host){
    // synchronize parameters between cpu memory and gpu memory
    float* devF = axis == 0? devP_: devQ_;
    float* hostF = axis == 0? hostP_: hostQ_;
    int size = next_x - start_x;
    if (device_to_host){
        CHECK_CUDA(hipMemcpy(hostF + (start_x * vdim_), devF + (start_x * vdim_),
                    sizeof(float)*size*vdim_, 
                   hipMemcpyDeviceToHost));
    } else{
        CHECK_CUDA(hipMemcpy(devF + (start_x * vdim_), hostF + (start_x * vdim_),
                    sizeof(float)*size*vdim_, 
                   hipMemcpyHostToDevice));
    }
    CHECK_CUDA(hipDeviceSynchronize());
}

int CuALS::get_vdim(){
    return vdim_;
}

std::pair<double, double> CuALS::partial_update(int start_x, 
        int next_x,
        int64_t* indptr,
        int* keys,
        float* vals,
        int axis){
    int thread_cnt = vdim_;
    size_t shared_memory_size = sizeof(float) * (3 * vdim_);
    int rows = axis == 0? P_rows_: Q_rows_;
    int op_rows = axis == 0? Q_rows_: P_rows_;
    float* P = axis == 0? devP_: devQ_;
    float* Q = axis == 0? devQ_: devP_;
    float reg = axis == 0? reg_u_: reg_i_;
    int64_t* _indptr = axis == 0?  lindptr_: rindptr_; 

    
    // copy data to gpu memory
    size_t beg = start_x == 0? 0: indptr[start_x - 1];
    size_t end = indptr[next_x - 1];
    CHECK_CUDA(hipMemcpy(keys_, keys, sizeof(int)*(end-beg), 
               hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(vals_, vals, sizeof(float)*(end-beg), 
               hipMemcpyHostToDevice));

    // set zeros for measuring losses
    if (compute_loss_){
        for (size_t i=0; i<block_cnt_; ++i){
            hostLossNume_[i] = 0;
            hostLossDeno_[i] = 0;
        }
        CHECK_CUDA(hipMemcpy(devLossNume_, hostLossDeno_, sizeof(float)*block_cnt_, 
                   hipMemcpyHostToDevice));
        CHECK_CUDA(hipMemcpy(devLossDeno_, hostLossDeno_, sizeof(float)*block_cnt_, 
                   hipMemcpyHostToDevice));
        
    } 
    CHECK_CUDA(hipDeviceSynchronize());
    

    // compute least square
    least_squares_cg_kernel<<<block_cnt_, thread_cnt, shared_memory_size>>>(
            dim_, vdim_, rows, op_rows, P, Q, devFF_, devLossNume_, devLossDeno_, 
            start_x, next_x, _indptr, keys_, vals_, alpha_, reg, adaptive_reg_,
            cg_tolerance_, num_cg_max_iters_, compute_loss_, eps_, axis);
    CHECK_CUDA(hipDeviceSynchronize());
    
   
    // accumulate losses
    double loss_nume = 0, loss_deno = 0;
    if (compute_loss_){
        CHECK_CUDA(hipMemcpy(hostLossNume_, devLossNume_, sizeof(float)*block_cnt_, 
                   hipMemcpyDeviceToHost));
        CHECK_CUDA(hipMemcpy(hostLossDeno_, devLossDeno_, sizeof(float)*block_cnt_, 
                   hipMemcpyDeviceToHost));
        CHECK_CUDA(hipDeviceSynchronize());
        for (size_t i=0; i<block_cnt_; ++i){
            loss_nume += hostLossNume_[i];
            loss_deno += hostLossDeno_[i];
        }
    }

    _synchronize(start_x, next_x, axis, true);

    return std::make_pair(loss_nume, loss_deno);
}

} // namespace cuda_als

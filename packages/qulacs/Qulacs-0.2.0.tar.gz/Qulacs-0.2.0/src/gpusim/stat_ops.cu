#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
//#include <hip/hip_runtime.h>

#include <cstdio>
#include <cmath>
#include <cstdlib>
#include <cstring>
#include <complex>
//#include <sys/time.h>

#include <limits.h>
#include <hipblas.h>
#include <hip/hip_complex.h>
#include "util_type.h"
#include "util_func.h"
#include "util.cuh"
#include "update_ops_cuda.h"
#include "stat_ops.h"
#include "stat_ops_device_functions.h"

__constant__ GTYPE matrix_const_gpu[4];
__constant__ unsigned int num_pauli_op_gpu[4];
__constant__ ITYPE bit_mask_gpu[4];
__constant__ GTYPE PHASE_90ROT_gpu[4];

__device__ double atomicAdd_double(double* address, double val)
{
	unsigned long long int* address_as_ull = (unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;
	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
	} while (assumed != old);
	return __longlong_as_double(old);
}

inline __device__ double __shfl_down_double(double var, unsigned int srcLane, int width = 32) {
	int2 a = *reinterpret_cast<int2*>(&var);
	a.x = __shfl_down_sync(0xffffffff, a.x, srcLane, width);
	a.y = __shfl_down_sync(0xffffffff, a.y, srcLane, width);
	return *reinterpret_cast<double*>(&a);
}

inline __device__ double __shfl_xor_double(double var, unsigned int srcLane, int width = 32) {
	int2 a = *reinterpret_cast<int2*>(&var);
	a.x = __shfl_xor_sync(0xffffffff, a.x, srcLane, width);
	a.y = __shfl_xor_sync(0xffffffff, a.y, srcLane, width);
	return *reinterpret_cast<double*>(&a);
}

inline __device__ double warpReduceSum_double(double val) {
#pragma unroll
    for (int offset = (warpSize >> 1); offset > 0; offset >>= 1)
        val += __shfl_down_sync(0xffffffff, val, offset);
	return val;
}

inline __device__ double warpAllReduceSum_double(double val){
    val += __shfl_xor_sync(0xffffffff, val, 16);
    val += __shfl_xor_sync(0xffffffff, val, 8);
    val += __shfl_xor_sync(0xffffffff, val, 4);
    val += __shfl_xor_sync(0xffffffff, val, 2);
    val += __shfl_xor_sync(0xffffffff, val, 1);
    return val;
}

__global__ void state_norm_squared_gpu(double* ret, GTYPE *state, ITYPE dim){
    double sum = double(0.0);
	GTYPE tmp;
    ITYPE idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (ITYPE i = idx; i < dim; i += blockDim.x * gridDim.x) {
		tmp = state[i];
        sum += tmp.x * tmp.x + tmp.y * tmp.y;
    }
	sum = warpReduceSum_double(sum);
	
	if ((threadIdx.x & (warpSize - 1)) == 0){
        atomicAdd_double(ret, sum);
    }
}

__host__ double state_norm_squared_cublas_host(void *state, ITYPE dim) {
    hipblasStatus_t status;
    hipblasHandle_t handle;
    double norm;
	GTYPE* state_gpu = reinterpret_cast<GTYPE*>(state);

    /* Initialize CUBLAS */
    status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS){
        fprintf(stderr, "!!!! CUBLAS initialization error\n");
        return EXIT_FAILURE;
    }

    status = hipblasDznrm2(handle, dim, state_gpu, 1, &norm);
	if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! hipblasDznrm2 execution error.\n");
        return EXIT_FAILURE;
    }

    /* Shutdown */
    status = hipblasDestroy(handle);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! shutdown error\n");
        return EXIT_FAILURE;
    }
 
	state = reinterpret_cast<void*>(state_gpu);
    return norm;
}

__host__ double state_norm_squared_host(void *state, ITYPE dim, void* stream, unsigned int device_number) {
	int current_device = get_current_device();
	if (device_number != current_device) hipSetDevice(device_number);

	hipStream_t* hip_stream = reinterpret_cast<hipStream_t*>(stream);
	hipError_t cudaStatus;
	double norm = 0.0;
	double* norm_gpu;
	GTYPE* state_gpu = reinterpret_cast<GTYPE*>(state);

	checkCudaErrors(hipMalloc((void**)&norm_gpu, sizeof(double)), __FILE__, __LINE__);
	checkCudaErrors(hipMemsetAsync(norm_gpu, 0, sizeof(double), *hip_stream), __FILE__, __LINE__);

	ITYPE loop_dim;
	if (dim <= 32) loop_dim = dim;
	else if (dim <= 4096) loop_dim = dim >> 2;
	else loop_dim = dim >> 5;

	unsigned int block = loop_dim <= 256 ? loop_dim : 256;
	unsigned int grid = loop_dim / block;

	state_norm_squared_gpu << < grid, block, 0, *hip_stream >> > (norm_gpu, state_gpu, dim);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();

	checkCudaErrors(cudaStatus, __FILE__, __LINE__);
	checkCudaErrors(hipStreamSynchronize(*hip_stream), __FILE__, __LINE__);
	checkCudaErrors(hipMemcpyAsync(&norm, norm_gpu, sizeof(double), hipMemcpyDeviceToHost, *hip_stream), __FILE__, __LINE__);

	checkCudaErrors(hipFree(norm_gpu), __FILE__, __LINE__);
	state = reinterpret_cast<void*>(state_gpu);
	//stream = reinterpret_cast<void*>(hip_stream);
	return norm;
}

__global__ void measurement_distribution_entropy_gpu(double* ret, const GTYPE *state, ITYPE dim){
	double sum = 0;
    const double eps = 1e-15;
	
    double prob;
    for (ITYPE index = blockIdx.x * blockDim.x + threadIdx.x; index < dim; index += blockDim.x * gridDim.x) {
		prob = hipCabs(state[index]);
        prob = prob * prob;
        if(prob > eps){
            sum += -1.0*prob*log(prob);
        } 
	}
	sum = warpReduceSum_double(sum);
	if ((threadIdx.x & (warpSize - 1)) == 0){
		atomicAdd_double(&(ret[0]), sum);
	}
}

__host__ double measurement_distribution_entropy_host(void* state, ITYPE dim, void* stream, unsigned int device_number) {
	int current_device = get_current_device();
	if (device_number != current_device) hipSetDevice(device_number);

	hipStream_t* hip_stream = reinterpret_cast<hipStream_t*>(stream);
	hipError_t cudaStatus;
	double ent;
	double* ent_gpu;
	GTYPE* state_gpu = reinterpret_cast<GTYPE*>(state);

	checkCudaErrors(hipMalloc((void**)&ent_gpu, sizeof(double)), __FILE__, __LINE__);
	checkCudaErrors(hipMemsetAsync(ent_gpu, 0, sizeof(double), *hip_stream), __FILE__, __LINE__);

	ITYPE loop_dim;
	if (dim <= 32) loop_dim = dim;
	else if (dim <= 4096) loop_dim = dim >> 2;
	else loop_dim = dim >> 5;

	unsigned int block = loop_dim <= 256 ? loop_dim : 256;
	unsigned int grid = loop_dim / block;

	measurement_distribution_entropy_gpu << <grid, block, 0, *hip_stream >> > (ent_gpu, state_gpu, dim);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();

	checkCudaErrors(cudaStatus, __FILE__, __LINE__);
	checkCudaErrors(hipStreamSynchronize(*hip_stream), __FILE__, __LINE__);
	checkCudaErrors(hipMemcpyAsync(&ent, ent_gpu, sizeof(double), hipMemcpyDeviceToHost, *hip_stream), __FILE__, __LINE__);

	checkCudaErrors(hipFree(ent_gpu), __FILE__, __LINE__);
	state = reinterpret_cast<void*>(state_gpu);
	//stream = reinterpret_cast<void*>(hip_stream);

	return ent;
}

__global__ void state_add_gpu(const GTYPE *state_added, GTYPE *state, ITYPE dim) {
    ITYPE state_index = blockIdx.x * blockDim.x + threadIdx.x;
	
    // loop varaibles
	const ITYPE loop_dim = dim;
	if(state_index<loop_dim){
		state[state_index] = hipCadd(state[state_index], state_added[state_index]);
    }
}

__host__ void state_add_host(void *state_added, void *state, ITYPE dim, void* stream, unsigned int device_number) {
	int current_device = get_current_device();
	if (device_number != current_device) hipSetDevice(device_number);

	GTYPE* state_gpu = reinterpret_cast<GTYPE*>(state);
	GTYPE* state_added_gpu = reinterpret_cast<GTYPE*>(state_added);
	hipStream_t* hip_stream = reinterpret_cast<hipStream_t*>(stream);

	ITYPE loop_dim = dim;

	unsigned int block = loop_dim <= 1024 ? loop_dim : 1024;
	unsigned int grid = loop_dim / block;

	state_add_gpu << <grid, block, 0, *hip_stream >> > (state_added_gpu, state_gpu, dim);

	checkCudaErrors(hipGetLastError(), __FILE__, __LINE__);
	checkCudaErrors(hipStreamSynchronize(*hip_stream), __FILE__, __LINE__);
	state = reinterpret_cast<void*>(state_gpu);
	state_added = reinterpret_cast<void*>(state_added_gpu);
	//stream = reinterpret_cast<void*>(hip_stream);

}

__global__ void state_multiply_gpu(const GTYPE coef, GTYPE *state, ITYPE dim) {
    ITYPE state_index = blockIdx.x * blockDim.x + threadIdx.x;
	
	const ITYPE loop_dim = dim;
	if(state_index<loop_dim){
		state[state_index] = hipCmul(state[state_index], coef);
	}
}

__host__ void state_multiply_host(CPPCTYPE coef, void *state, ITYPE dim, void* stream, unsigned int device_number) {
	int current_device = get_current_device();
	if (device_number != current_device) hipSetDevice(device_number);

	GTYPE* state_gpu = reinterpret_cast<GTYPE*>(state);
	hipStream_t* hip_stream = reinterpret_cast<hipStream_t*>(stream);
	ITYPE loop_dim = dim;

	GTYPE coef_gpu = make_hipDoubleComplex(coef.real(), coef.imag());
	unsigned int block = loop_dim <= 1024 ? loop_dim : 1024;
	unsigned int grid = loop_dim / block;

	state_multiply_gpu << <grid, block, 0, *hip_stream >> > (coef_gpu, state_gpu, dim);

	checkCudaErrors(hipGetLastError(), __FILE__, __LINE__);
	checkCudaErrors(hipStreamSynchronize(*hip_stream), __FILE__, __LINE__);
	state = reinterpret_cast<void*>(state_gpu);
	//stream = reinterpret_cast<void*>(hip_stream);
}

__global__ void inner_product_gpu(GTYPE *ret, const GTYPE *psi, const GTYPE *phi, ITYPE dim){
	GTYPE sum = make_hipDoubleComplex(0.0, 0.0);
	for (ITYPE i = blockIdx.x * blockDim.x + threadIdx.x; i < dim; i += blockDim.x * gridDim.x) {
		sum = hipCadd(sum, hipCmul(hipConj(psi[i]), phi[i]));
	}
	sum.x = warpReduceSum_double(sum.x);
	sum.y = warpReduceSum_double(sum.y);
	if ((threadIdx.x & (warpSize - 1)) == 0){
		atomicAdd_double(&(ret[0].x), sum.x);
		atomicAdd_double(&(ret[0].y), sum.y);
	}
}

/*
__host__ CPPCTYPE inner_product_cublas_host(const void *bra_state, const void *ket_state, ITYPE dim) {
	const GTYPE* bra_state_gpu = reinterpret_cast<const GTYPE*>(bra_state);
	const GTYPE* ket_state_gpu = reinterpret_cast<const GTYPE*>(ket_state);
    hipblasStatus_t status;
    hipblasHandle_t handle;
	GTYPE ret_g;
    CPPCTYPE ret;

    status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS){
        fprintf(stderr, "!!!! CUBLAS initialization error\n");
        return EXIT_FAILURE;
    }

	status = hipblasZdotc(handle, dim, bra_state_gpu, 1, ket_state_gpu, 1, &ret_g);
	if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! cublasZDotc execution error.\n");
        return EXIT_FAILURE;
    }

    status = hipblasDestroy(handle);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! shutdown error\n");
        return EXIT_FAILURE;
    }

	bra_state = reinterpret_cast<const void*>(bra_state_gpu);
	ket_state = reinterpret_cast<const void*>(ket_state_gpu);
    ret = CPPCTYPE(hipCreal(ret_g), hipCimag(ret_g));
	return ret;
}
*/

__host__ CPPCTYPE inner_product_cublas_host(const void* bra_state, const void* ket_state, ITYPE dim, void* stream, unsigned int device_number) {
	int current_device = get_current_device();
	if (device_number != current_device) hipSetDevice(device_number);

	hipStream_t* hip_stream = reinterpret_cast<hipStream_t*>(stream);
	const GTYPE* bra_state_gpu = reinterpret_cast<const GTYPE*>(bra_state);
	const GTYPE* ket_state_gpu = reinterpret_cast<const GTYPE*>(ket_state);
	hipblasStatus_t status;
	hipblasHandle_t handle;
	GTYPE ret_g;
	CPPCTYPE ret;

	/* Initialize CUBLAS */
	status = hipblasCreate(&handle);
	if (status != HIPBLAS_STATUS_SUCCESS) {
		fprintf(stderr, "!!!! CUBLAS initialization error\n");
		return EXIT_FAILURE;
	}

	status = hipblasSetStream(handle, *hip_stream);
	if (status != HIPBLAS_STATUS_SUCCESS) {
		fprintf(stderr, "!!!! set cublas to cuda stream error\n");
		return EXIT_FAILURE;
	}

	status = hipblasZdotc(handle, dim, bra_state_gpu, 1, ket_state_gpu, 1, &ret_g);
	if (status != HIPBLAS_STATUS_SUCCESS) {
		fprintf(stderr, "!!!! cublasZDotc execution error.\n");
		return EXIT_FAILURE;
	}

	/* Shutdown */
	status = hipblasDestroy(handle);

	if (status != HIPBLAS_STATUS_SUCCESS) {
		fprintf(stderr, "!!!! shutdown error\n");
		return EXIT_FAILURE;
	}

	bra_state = reinterpret_cast<const void*>(bra_state_gpu);
	ket_state = reinterpret_cast<const void*>(ket_state_gpu);
	//stream = reinterpret_cast<void*>(hip_stream);
	ret = CPPCTYPE(hipCreal(ret_g), hipCimag(ret_g));
	return ret;
}

__host__ CPPCTYPE inner_product_original_host(const void *bra_state, const void *ket_state, ITYPE dim, void* stream, unsigned int device_number) {
	int current_device = get_current_device();
	if (device_number != current_device) hipSetDevice(device_number);

	const GTYPE* bra_state_gpu = reinterpret_cast<const GTYPE*>(bra_state);
	const GTYPE* ket_state_gpu = reinterpret_cast<const GTYPE*>(ket_state);
	hipStream_t* hip_stream = reinterpret_cast<hipStream_t*>(stream);
	hipError_t cudaStatus;
	CPPCTYPE ret = CPPCTYPE(0.0, 0.0);
	GTYPE *ret_gpu;

	checkCudaErrors(hipMalloc((void**)&ret_gpu, sizeof(GTYPE)), __FILE__, __LINE__);
	checkCudaErrors(hipMemcpyAsync(ret_gpu, &ret, sizeof(GTYPE), hipMemcpyHostToDevice, *hip_stream), __FILE__, __LINE__);

	ITYPE loop_dim;
	if (dim <= 32) loop_dim = dim;
	else if (dim <= 4096) loop_dim = dim >> 2;
	else loop_dim = dim >> 5;

	unsigned int block = loop_dim <= 256 ? loop_dim : 256;
	unsigned int grid = loop_dim / block;

	inner_product_gpu << <grid, block, 0, *hip_stream >> > (ret_gpu, bra_state_gpu, ket_state_gpu, dim);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();

	checkCudaErrors(cudaStatus, __FILE__, __LINE__);
	checkCudaErrors(hipStreamSynchronize(*hip_stream), __FILE__, __LINE__);
	checkCudaErrors(hipMemcpyAsync(&ret, ret_gpu, sizeof(GTYPE), hipMemcpyDeviceToHost, *hip_stream), __FILE__, __LINE__);

	checkCudaErrors(hipFree(ret_gpu), __FILE__, __LINE__);
	bra_state = reinterpret_cast<const void*>(bra_state_gpu);
	ket_state = reinterpret_cast<const void*>(ket_state_gpu);
	//stream = reinterpret_cast<void*>(hip_stream);
	return ret;
}

__host__ CPPCTYPE inner_product_host(const void *bra_state, const void *ket_state, ITYPE dim, void* stream, unsigned int device_number){
	if (dim <= INT_MAX) {
		// ���Ƃ�cublas�ł��g���悤�ɒ���
		return inner_product_original_host(bra_state, ket_state, dim, stream, device_number);
		//return inner_product_cublas_host(bra_state, ket_state, dim, stream, device_number);
	}
	else {
		return inner_product_original_host(bra_state, ket_state, dim, stream, device_number);
	}
}

__global__ void expectation_value_PauliI_gpu(double *ret, GTYPE *state, unsigned int target_qubit_index, ITYPE dim){
    double sum=0.0;
    ITYPE loop_dim = dim;
    GTYPE tmp_state;
	for (ITYPE state_index = blockIdx.x * blockDim.x + threadIdx.x; state_index < loop_dim; state_index += blockDim.x * gridDim.x) {
        tmp_state=state[state_index];
        sum += hipCreal( hipCmul( hipConj(tmp_state), tmp_state ) );
	}
	sum = warpReduceSum_double(sum);
	if ((threadIdx.x & (warpSize - 1)) == 0){
		atomicAdd_double(&(ret[0]), sum);
	}
}

__global__ void expectation_value_PauliX_gpu(double *ret, GTYPE *state, unsigned int target_qubit_index, ITYPE dim){
    double sum = 0.0;
    ITYPE basis0, basis1;
    ITYPE loop_dim = dim>>1;
	for (ITYPE state_index = blockIdx.x * blockDim.x + threadIdx.x; state_index < loop_dim; state_index += blockDim.x * gridDim.x) {
        basis0 = (state_index >> target_qubit_index);
        basis0 = basis0 << (target_qubit_index + 1);
        basis0 += state_index & ((1ULL << target_qubit_index) - 1);
        basis1 = basis0 ^ (1ULL << target_qubit_index);

        sum += hipCreal( hipCmul(hipConj(state[basis0]), state[basis1]) );
	}
    sum*=2;
	sum = warpReduceSum_double(sum);
	if ((threadIdx.x & (warpSize - 1)) == 0){
		atomicAdd_double(&(ret[0]), sum);
    }
}

__global__ void expectation_value_PauliY_gpu(double *ret, GTYPE *state, unsigned int target_qubit_index, ITYPE dim){
	double sum = 0.0;
    ITYPE basis0, basis1;
    ITYPE loop_dim = dim>>1;
	for (ITYPE state_index = blockIdx.x * blockDim.x + threadIdx.x; state_index < loop_dim; state_index += blockDim.x * gridDim.x) {
        basis0 = (state_index >> target_qubit_index);
        basis0 = basis0 << (target_qubit_index + 1);
        basis0 += state_index & ((1ULL << target_qubit_index) - 1);
        basis1 = basis0 ^ (1ULL << target_qubit_index);
        sum += hipCimag( hipCmul(hipConj(state[basis0]), state[basis1]) );
	}
    sum*=2;
	sum = warpReduceSum_double(sum);
	if ((threadIdx.x & (warpSize - 1)) == 0){
		atomicAdd_double(&(ret[0]), sum);
    }
}

__global__ void expectation_value_PauliZ_gpu(double *ret, GTYPE *state, unsigned int target_qubit_index, ITYPE dim){
    double sum=0.0;
    ITYPE basis0, basis1;
    ITYPE loop_dim = dim>>1;
	for (ITYPE state_index = blockIdx.x * blockDim.x + threadIdx.x; state_index < loop_dim; state_index += blockDim.x * gridDim.x) {
        basis0 = (state_index >> target_qubit_index);
        basis0 = basis0 << (target_qubit_index + 1);
        basis0 += state_index & ((1ULL << target_qubit_index) - 1);
        basis1 = basis0 ^ (1ULL << target_qubit_index);
        sum += hipCreal( hipCmul( hipConj(state[basis0]), state[basis0]) )
            -  hipCreal( hipCmul( hipConj(state[basis1]), state[basis1]) );
	}
	sum = warpReduceSum_double(sum);
	if ((threadIdx.x & (warpSize - 1)) == 0){
		atomicAdd_double(&(ret[0]), sum);
	}
}

__host__ double expectation_value_single_qubit_Pauli_operator_host(unsigned int operator_index, unsigned int target_qubit_index, void* state, ITYPE dim, void* stream, unsigned int device_number) {
	int current_device = get_current_device();
	if (device_number != current_device) hipSetDevice(device_number);

	GTYPE* state_gpu = reinterpret_cast<GTYPE*>(state);
	hipStream_t* hip_stream = reinterpret_cast<hipStream_t*>(stream);
	double h_ret = 0.0;
	double* d_ret;

	// this loop_dim is not the same as that of the gpu function
	// and the function uses grid stride loops
	ITYPE loop_dim;
	if (dim <= 64) loop_dim = dim >> 1;
	else if (dim <= (1ULL << 11)) loop_dim = dim >> 2;
	else loop_dim = dim >> 5;


	unsigned int block = loop_dim <= 256 ? loop_dim : 256;
	unsigned int grid = loop_dim / block;

	checkCudaErrors(hipMalloc((void**)&d_ret, sizeof(double)), __FILE__, __LINE__);
	checkCudaErrors(hipMemsetAsync(d_ret, 0, sizeof(double), *hip_stream), __FILE__, __LINE__);

	if (operator_index == 1) {
		expectation_value_PauliX_gpu << <grid, block, 0, *hip_stream >> > (d_ret, state_gpu, target_qubit_index, dim);
	}
	else if (operator_index == 2) {
		expectation_value_PauliY_gpu << <grid, block, 0, *hip_stream >> > (d_ret, state_gpu, target_qubit_index, dim);
	}
	else if (operator_index == 3) {
		expectation_value_PauliZ_gpu << <grid, block, 0, *hip_stream >> > (d_ret, state_gpu, target_qubit_index, dim);
	}
	else if (operator_index == 0) {
		expectation_value_PauliI_gpu << <grid, block, 0, *hip_stream >> > (d_ret, state_gpu, target_qubit_index, dim);
	}
	else {
		printf("operator_index must be an integer of 0, 1, 2, or 3!!");
	}

	checkCudaErrors(hipStreamSynchronize(*hip_stream), __FILE__, __LINE__);
	checkCudaErrors(hipMemcpyAsync(&h_ret, d_ret, sizeof(double), hipMemcpyDeviceToHost, *hip_stream), __FILE__, __LINE__);
	checkCudaErrors(hipFree(d_ret), __FILE__, __LINE__);
	state = reinterpret_cast<void*>(state_gpu);
	//stream = reinterpret_cast<void*>(hip_stream);
	return h_ret;
}

__device__ void multi_Z_gate_device(ITYPE bit_mask, ITYPE DIM, GTYPE *psi_gpu)
{
	ITYPE idx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int minus_cnt = 0;
	if (idx < DIM){
		minus_cnt = __popcll(idx&bit_mask);
		if (minus_cnt & 1) psi_gpu[idx] = make_hipDoubleComplex(-psi_gpu[idx].x, -psi_gpu[idx].y);
	}
}

__global__ void multi_Z_gate_gpu(ITYPE bit_mask, ITYPE DIM, GTYPE *psi_gpu)
{
	multi_Z_gate_device(bit_mask, DIM, psi_gpu);
}

__host__ void multi_Z_gate_host(int* gates, void* state, ITYPE dim, int n_qubits, void* stream, unsigned int device_number) {
	int current_device = get_current_device();
	if (device_number != current_device) hipSetDevice(device_number);

	GTYPE* state_gpu = reinterpret_cast<GTYPE*>(state);
	hipStream_t* hip_stream = reinterpret_cast<hipStream_t*>(stream);
	ITYPE bit_mask = 0;
	for (int i = 0; i < n_qubits; ++i) {
		if (gates[i] == 3) bit_mask ^= (1ULL << i);
	}
	hipError_t cudaStatus;
	unsigned int block = dim <= 1024 ? dim : 1024;
	unsigned int grid = dim / block;
	multi_Z_gate_gpu << <grid, block, 0, *hip_stream >> > (bit_mask, dim, state_gpu);
	checkCudaErrors(hipStreamSynchronize(*hip_stream), __FILE__, __LINE__);
	cudaStatus = hipGetLastError();
	checkCudaErrors(cudaStatus, __FILE__, __LINE__);
	state = reinterpret_cast<void*>(state_gpu);
	//stream = reinterpret_cast<void*>(hip_stream);
}

__device__ GTYPE multi_Z_get_expectation_value_device(ITYPE idx, ITYPE bit_mask, ITYPE dim, GTYPE *psi_gpu)
{
	GTYPE ret=make_hipDoubleComplex(0.0,0.0);
	// ITYPE idx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int minus_cnt = 0;
	if (idx < dim){
		GTYPE tmp_psi_gpu = psi_gpu[idx];
		minus_cnt = __popcll(idx&bit_mask);
		if (minus_cnt & 1) tmp_psi_gpu = make_hipDoubleComplex(-tmp_psi_gpu.x, -tmp_psi_gpu.y);
		ret = hipCmul(hipConj(psi_gpu[idx]), tmp_psi_gpu);
	}
	return ret;
}

__global__ void multi_Z_get_expectation_value_gpu(GTYPE *ret, ITYPE bit_mask, ITYPE DIM, GTYPE *psi_gpu)
{
	GTYPE sum = make_hipDoubleComplex(0.0, 0.0);
	GTYPE tmp;
	for (ITYPE state = blockIdx.x * blockDim.x + threadIdx.x; state < DIM; state += blockDim.x * gridDim.x) {
		tmp = multi_Z_get_expectation_value_device(state, bit_mask, DIM, psi_gpu);
		sum = hipCadd(sum, tmp);
	}
	sum.x = warpReduceSum_double(sum.x);
	sum.y = warpReduceSum_double(sum.y);
	if ((threadIdx.x & (warpSize - 1)) == 0){
		atomicAdd_double(&(ret[0].x), sum.x);
		atomicAdd_double(&(ret[0].y), sum.y);
	}
}

__device__ GTYPE multipauli_get_expectation_value_device(ITYPE idx, ITYPE DIM, GTYPE *psi_gpu, int n_qubits){
	GTYPE ret;
	GTYPE tmp_psi, tmp_prev_state_psi, tmp_state_psi;
	ITYPE prev_state, state;
	int num_y1 = 0;
	int num_z1 = 0;
	int i_cnt = 0;
	int minus_cnt = 0;
	if (idx < DIM){
		i_cnt = num_pauli_op_gpu[2];
		num_y1 = __popcll(idx&bit_mask_gpu[2]);
		num_z1 = __popcll(idx&bit_mask_gpu[3]);
		minus_cnt = num_y1 + num_z1;
		prev_state = idx;
		state = idx^(bit_mask_gpu[1]+bit_mask_gpu[2]);
		tmp_prev_state_psi = psi_gpu[prev_state];
		tmp_state_psi = psi_gpu[state];
		// swap
		tmp_psi = tmp_state_psi;
		tmp_state_psi = tmp_prev_state_psi;
		tmp_prev_state_psi = tmp_psi;
		if (minus_cnt & 1) tmp_state_psi = make_hipDoubleComplex(-tmp_state_psi.x, -tmp_state_psi.y);
		if (i_cnt & 1) tmp_state_psi = make_hipDoubleComplex(tmp_state_psi.y, tmp_state_psi.x);
		if ((i_cnt >> 1) & 1) tmp_state_psi = make_hipDoubleComplex(-tmp_state_psi.x, -tmp_state_psi.y);
		// tmp_state      -> state      : state*conj(tmp_state)
		// tmp_prev_state -> prev_state : prev_state*conj(tmp_prev_state)
		ret = hipCmul(tmp_state_psi, hipConj(psi_gpu[state]));
	}
	return ret;
}

__global__ void multipauli_get_expectation_value_gpu(GTYPE* ret, ITYPE DIM, GTYPE *psi_gpu, int n_qubits){
	GTYPE sum = make_hipDoubleComplex(0.0, 0.0);
	GTYPE tmp;
	for (ITYPE state = blockIdx.x * blockDim.x + threadIdx.x; state < DIM; state += blockDim.x * gridDim.x) {
		tmp = multipauli_get_expectation_value_device(state, DIM, psi_gpu, n_qubits);
		sum = hipCadd(sum, tmp);
	}
	sum.x = warpReduceSum_double(sum.x);
	sum.y = warpReduceSum_double(sum.y);
	if ((threadIdx.x & (warpSize - 1)) == 0){
		atomicAdd_double(&(ret[0].x), sum.x);
		atomicAdd_double(&(ret[0].y), sum.y);
	}
}

__host__ double multipauli_get_expectation_value_host(unsigned int* gates, void* state, ITYPE dim, int n_qubits, void* stream, unsigned int device_number) {
	int current_device = get_current_device();
	if (device_number != current_device) hipSetDevice(device_number);

	GTYPE* state_gpu = reinterpret_cast<GTYPE*>(state);
	hipStream_t* hip_stream = reinterpret_cast<hipStream_t*>(stream);
	CPPCTYPE ret[1];
	ret[0] = CPPCTYPE(0, 0);
	GTYPE *ret_gpu;

	checkCudaErrors(hipMalloc((void**)&ret_gpu, sizeof(GTYPE)), __FILE__, __LINE__);
	checkCudaErrors(hipMemsetAsync(ret_gpu, 0, sizeof(double), *hip_stream), __FILE__, __LINE__);

	ITYPE loop_dim;
	if (dim <= 32) loop_dim = dim >> 1;
	else if (dim <= (1ULL << 11)) loop_dim = dim >> 2;
	else loop_dim = dim >> 5;

	unsigned int block = loop_dim <= 256 ? loop_dim : 256;
	unsigned int grid = loop_dim / block;

	unsigned int num_pauli_op[4] = { 0, 0, 0, 0 };
	for (int i = 0; i < n_qubits; ++i) ++num_pauli_op[gates[i]];
	ITYPE bit_mask[4] = { 0, 0, 0, 0 };
	for (int i = 0; i < n_qubits; ++i) {
		bit_mask[gates[i]] ^= (1ULL << i);
	}
	if (num_pauli_op[1] == 0 && num_pauli_op[2] == 0) {
		multi_Z_get_expectation_value_gpu << <grid, block, 0, *hip_stream >> > (ret_gpu, bit_mask[3], dim, state_gpu);
		checkCudaErrors(hipStreamSynchronize(*hip_stream), __FILE__, __LINE__);
		checkCudaErrors(hipGetLastError(), __FILE__, __LINE__);
		checkCudaErrors(hipMemcpyAsync(ret, ret_gpu, sizeof(CPPCTYPE), hipMemcpyDeviceToHost, *hip_stream), __FILE__, __LINE__);
		checkCudaErrors(hipFree(ret_gpu), __FILE__, __LINE__);
		state = reinterpret_cast<void*>(state_gpu);
		return ret[0].real();
	}

	checkCudaErrors(hipMemcpyToSymbolAsync(HIP_SYMBOL(num_pauli_op_gpu), num_pauli_op, sizeof(unsigned int) * 4, 0, hipMemcpyHostToDevice, *hip_stream), __FILE__, __LINE__);
	checkCudaErrors(hipMemcpyToSymbolAsync(HIP_SYMBOL(bit_mask_gpu), bit_mask, sizeof(ITYPE) * 4, 0, hipMemcpyHostToDevice, *hip_stream), __FILE__, __LINE__);

	multipauli_get_expectation_value_gpu << <grid, block, 0, *hip_stream >> > (ret_gpu, dim, state_gpu, n_qubits);

	checkCudaErrors(hipStreamSynchronize(*hip_stream), __FILE__, __LINE__);
	checkCudaErrors(hipGetLastError(), __FILE__, __LINE__);
	checkCudaErrors(hipMemcpyAsync(ret, ret_gpu, sizeof(CPPCTYPE), hipMemcpyDeviceToHost, *hip_stream), __FILE__, __LINE__);
	checkCudaErrors(hipFree(ret_gpu), __FILE__, __LINE__);
	state = reinterpret_cast<void*>(state_gpu);
	//stream = reinterpret_cast<void*>(hip_stream);
	return ret[0].real();
}

// calculate probability with which we obtain 0 at target qubit
__global__ void M0_prob_gpu(double* ret, UINT target_qubit_index, const GTYPE* state, ITYPE dim){
    const ITYPE loop_dim = dim>>1;
    double sum =0.;
    double tmp;
	for (ITYPE state_index = blockIdx.x * blockDim.x + threadIdx.x; state_index < loop_dim; state_index += blockDim.x * gridDim.x) {
        ITYPE basis_0 = insert_zero_to_basis_index_device(state_index, target_qubit_index);
        tmp = hipCabs(state[basis_0]);
        sum += tmp*tmp;
	}
	sum = warpReduceSum_double(sum);
	if ((threadIdx.x & (warpSize - 1)) == 0){
		atomicAdd_double(&(ret[0]), sum);
	}
}

// calculate probability with which we obtain 0 at target qubit
__host__ double M0_prob_host(UINT target_qubit_index, void* state, ITYPE dim, void* stream, unsigned int device_number) {
	int current_device = get_current_device();
	if (device_number != current_device) hipSetDevice(device_number);

	GTYPE* state_gpu = reinterpret_cast<GTYPE*>(state);
	hipStream_t* hip_stream = reinterpret_cast<hipStream_t*>(stream);
	double ret[1] = { 0.0 };
	double *ret_gpu;

	checkCudaErrors(hipMalloc((void**)&ret_gpu, sizeof(double)), __FILE__, __LINE__);
	checkCudaErrors(hipMemsetAsync(ret_gpu, 0, sizeof(double), *hip_stream), __FILE__, __LINE__);

	ITYPE loop_dim;

	if (dim <= 64) loop_dim = dim >> 1;
	else if (dim <= (1ULL << 11)) loop_dim = dim >> 2;
	else loop_dim = dim >> 5;

	unsigned int block = loop_dim <= 256 ? loop_dim : 256;
	unsigned int grid = loop_dim / block;

	M0_prob_gpu << <grid, block, 0, *hip_stream >> > (ret_gpu, target_qubit_index, state_gpu, dim);

	checkCudaErrors(hipStreamSynchronize(*hip_stream), __FILE__, __LINE__);
	checkCudaErrors(hipGetLastError(), __FILE__, __LINE__);
	checkCudaErrors(hipMemcpyAsync(ret, ret_gpu, sizeof(double), hipMemcpyDeviceToHost, *hip_stream), __FILE__, __LINE__);
	checkCudaErrors(hipFree(ret_gpu), __FILE__, __LINE__);
	state = reinterpret_cast<void*>(state_gpu);
	// stream = reinterpret_cast<void*>(hip_stream);
	return ret[0];
}

// calculate probability with which we obtain 1 at target qubit
__global__ void M1_prob_gpu(double* ret, UINT target_qubit_index, const GTYPE* state, ITYPE dim){
    const ITYPE loop_dim = dim>>1;
    const ITYPE mask = 1ULL << target_qubit_index;
    double sum =0.;
    double tmp;
	for (ITYPE state_index = blockIdx.x * blockDim.x + threadIdx.x; state_index < loop_dim; state_index += blockDim.x * gridDim.x) {
        ITYPE basis_1 = insert_zero_to_basis_index_device(state_index, target_qubit_index) ^ mask;
        tmp = hipCabs(state[basis_1]);
        sum += tmp*tmp;
	}
	sum = warpReduceSum_double(sum);
	if ((threadIdx.x & (warpSize - 1)) == 0){
		atomicAdd_double(&(ret[0]), sum);
	}
}

__host__ double M1_prob_host(UINT target_qubit_index, void* state, ITYPE dim, void* stream, unsigned int device_number) {
	int current_device = get_current_device();
	if (device_number != current_device) hipSetDevice(device_number);

	GTYPE* state_gpu = reinterpret_cast<GTYPE*>(state);
	hipStream_t* hip_stream = reinterpret_cast<hipStream_t*>(stream);
	double ret[1] = { 0.0 };
	double *ret_gpu;

	checkCudaErrors(hipMalloc((void**)&ret_gpu, sizeof(double)), __FILE__, __LINE__);
	checkCudaErrors(hipMemsetAsync(ret_gpu, 0, sizeof(double), *hip_stream), __FILE__, __LINE__);

	ITYPE loop_dim;

	if (dim <= 64) loop_dim = dim >> 1;
	else if (dim <= (1ULL << 11)) loop_dim = dim >> 2;
	else loop_dim = dim >> 5;

	unsigned int block = loop_dim <= 256 ? loop_dim : 256;
	unsigned int grid = loop_dim / block;

	M1_prob_gpu << <grid, block, 0, *hip_stream >> > (ret_gpu, target_qubit_index, state_gpu, dim);

	checkCudaErrors(hipStreamSynchronize(*hip_stream), __FILE__, __LINE__);
	checkCudaErrors(hipGetLastError(), __FILE__, __LINE__);
	checkCudaErrors(hipMemcpyAsync(ret, ret_gpu, sizeof(double), hipMemcpyDeviceToHost, *hip_stream), __FILE__, __LINE__);
	checkCudaErrors(hipFree(ret_gpu), __FILE__, __LINE__);
	state = reinterpret_cast<void*>(state_gpu);
	//stream = reinterpret_cast<void*>(hip_stream);
	return ret[0];
}

// calculate merginal probability with which we obtain the set of values measured_value_list at sorted_target_qubit_index_list
// warning: sorted_target_qubit_index_list must be sorted.
__global__ void marginal_prob_gpu(double* ret_gpu, const UINT* sorted_target_qubit_index_list, const UINT* measured_value_list, UINT target_qubit_index_count, const GTYPE* state, ITYPE dim){
    ITYPE loop_dim = dim >> target_qubit_index_count;
    double sum =0.;
    double tmp;
	for (ITYPE state_index = blockIdx.x * blockDim.x + threadIdx.x; state_index < loop_dim; state_index += blockDim.x * gridDim.x) {
        ITYPE basis = state_index;
        for(UINT cursor=0; cursor < target_qubit_index_count ; cursor++){
            UINT insert_index = sorted_target_qubit_index_list[cursor];
            ITYPE mask = 1ULL << insert_index;
            basis = insert_zero_to_basis_index_device(basis, insert_index );
            basis ^= mask * measured_value_list[cursor];
        }
        tmp = hipCabs(state[basis]);
        sum += tmp*tmp;
	}
	sum = warpReduceSum_double(sum);
	if ((threadIdx.x & (warpSize - 1)) == 0){
		atomicAdd_double(&(ret_gpu[0]), sum);
	}
}

__host__ double marginal_prob_host(UINT* sorted_target_qubit_index_list, UINT* measured_value_list, UINT target_qubit_index_count, void* state, ITYPE dim, void* stream, unsigned int device_number) {
	int current_device = get_current_device();
	if (device_number != current_device) hipSetDevice(device_number);

	GTYPE* state_gpu = reinterpret_cast<GTYPE*>(state);
	hipStream_t* hip_stream = reinterpret_cast<hipStream_t*>(stream);
	double ret[1] = { 0.0 };
	double *ret_gpu;
	UINT* sorted_target_qubit_index_list_gpu;
	UINT* measured_value_list_gpu;

	checkCudaErrors(hipMalloc((void**)&ret_gpu, sizeof(double)), __FILE__, __LINE__);
	checkCudaErrors(hipMemcpyAsync(ret_gpu, ret, sizeof(double), hipMemcpyHostToDevice, *hip_stream), __FILE__, __LINE__);
	checkCudaErrors(hipMalloc((void**)&sorted_target_qubit_index_list_gpu, sizeof(UINT)*target_qubit_index_count), __FILE__, __LINE__);
	checkCudaErrors(hipMemcpyAsync(sorted_target_qubit_index_list_gpu, sorted_target_qubit_index_list, sizeof(UINT)*target_qubit_index_count, hipMemcpyHostToDevice, *hip_stream), __FILE__, __LINE__);
	checkCudaErrors(hipMalloc((void**)&measured_value_list_gpu, sizeof(UINT)*target_qubit_index_count), __FILE__, __LINE__);
	checkCudaErrors(hipMemcpyAsync(measured_value_list_gpu, measured_value_list, sizeof(UINT)*target_qubit_index_count, hipMemcpyHostToDevice, *hip_stream), __FILE__, __LINE__);

	unsigned int block = dim <= 1024 ? dim : 1024;
	unsigned int grid = dim / block;

	marginal_prob_gpu << <grid, block, 0, *hip_stream >> > (ret_gpu, sorted_target_qubit_index_list_gpu, measured_value_list_gpu, target_qubit_index_count, state_gpu, dim);

	checkCudaErrors(hipStreamSynchronize(*hip_stream), __FILE__, __LINE__);
	checkCudaErrors(hipGetLastError(), __FILE__, __LINE__);
	checkCudaErrors(hipMemcpyAsync(ret, ret_gpu, sizeof(double), hipMemcpyDeviceToHost, *hip_stream), __FILE__, __LINE__);
	checkCudaErrors(hipFree(ret_gpu), __FILE__, __LINE__);
	checkCudaErrors(hipFree(sorted_target_qubit_index_list_gpu), __FILE__, __LINE__);
	checkCudaErrors(hipFree(measured_value_list_gpu), __FILE__, __LINE__);
	state = reinterpret_cast<void*>(state_gpu);
	//stream = reinterpret_cast<void*>(hip_stream);
	return ret[0];
}

__global__ void expectation_value_multi_qubit_Pauli_operator_XZ_mask_gpu(double* ret_gpu, ITYPE bit_flip_mask, ITYPE phase_flip_mask, UINT global_phase_90rot_count,UINT pivot_qubit_index, GTYPE* state, ITYPE dim){
    const ITYPE loop_dim = dim>>1;
    double sum = 0.;
    double tmp;
	for (ITYPE state_index = blockIdx.x * blockDim.x + threadIdx.x; state_index < loop_dim; state_index += blockDim.x * gridDim.x) {
        ITYPE basis_0 = insert_zero_to_basis_index_device(state_index, pivot_qubit_index);
        ITYPE basis_1 = basis_0 ^ bit_flip_mask;
        UINT sign_0 = __popcll(basis_0 & phase_flip_mask)&1;
        
        tmp = hipCreal(hipCmul( hipCmul(state[basis_0], hipConj(state[basis_1])), PHASE_90ROT_gpu[ (global_phase_90rot_count + sign_0*2)&3 ]))*2.0;
        sum += tmp;
    }
	sum = warpReduceSum_double(sum);
	if ((threadIdx.x & (warpSize - 1)) == 0){
		atomicAdd_double(&(ret_gpu[0]), sum);
	}
}

__host__ double expectation_value_multi_qubit_Pauli_operator_XZ_mask_host(ITYPE bit_flip_mask, ITYPE phase_flip_mask, UINT global_phase_90rot_count, UINT pivot_qubit_index, void* state, ITYPE dim, void* stream, unsigned int device_number) {
	int current_device = get_current_device();
	if (device_number != current_device) hipSetDevice(device_number);

	GTYPE* state_gpu = reinterpret_cast<GTYPE*>(state);
	hipStream_t* hip_stream = reinterpret_cast<hipStream_t*>(stream);
	hipError_t cudaStatus;
	double ret;
	double* ret_gpu;
	CPPCTYPE PHASE_90ROT[4] = {
		CPPCTYPE(1.0, 0.0),
		CPPCTYPE(0.0, 1.0),
		CPPCTYPE(-1.0,0.0),
		CPPCTYPE(0.0, -1.0) };

	checkCudaErrors(hipMalloc((void**)&ret_gpu, sizeof(double)), __FILE__, __LINE__);
	checkCudaErrors(hipMemsetAsync(ret_gpu, 0, sizeof(double), *hip_stream), __FILE__, __LINE__);
	checkCudaErrors(hipMemcpyToSymbolAsync(HIP_SYMBOL(PHASE_90ROT_gpu), PHASE_90ROT, sizeof(GTYPE) * 4, 0, hipMemcpyHostToDevice, *hip_stream), __FILE__, __LINE__);

	ITYPE loop_dim;

	if (dim <= 64) loop_dim = dim >> 1;
	else if (dim <= (1ULL << 11)) loop_dim = dim >> 2;
	else loop_dim = dim >> 5;

	unsigned int block = loop_dim <= 256 ? loop_dim : 256;
	unsigned int grid = loop_dim / block;

	expectation_value_multi_qubit_Pauli_operator_XZ_mask_gpu << <grid, block, 0, *hip_stream >> > (ret_gpu, bit_flip_mask, phase_flip_mask, global_phase_90rot_count, pivot_qubit_index, state_gpu, dim);

	checkCudaErrors(hipStreamSynchronize(*hip_stream), __FILE__, __LINE__);
	cudaStatus = hipGetLastError();
	checkCudaErrors(cudaStatus, __FILE__, __LINE__);
	checkCudaErrors(hipMemcpyAsync(&ret, ret_gpu, sizeof(double), hipMemcpyDeviceToHost, *hip_stream), __FILE__, __LINE__);
	checkCudaErrors(hipFree(ret_gpu), __FILE__, __LINE__);
	state = reinterpret_cast<void*>(state_gpu);
	// stream = reinterpret_cast<void*>(hip_stream);

	return ret;
}

__global__ void expectation_value_multi_qubit_Pauli_operator_Z_mask_gpu(double* ret_gpu, ITYPE phase_flip_mask, const GTYPE* state, ITYPE dim){
    const ITYPE loop_dim = dim;
    double sum = 0.;
    double tmp;
	for (ITYPE state_index = blockIdx.x * blockDim.x + threadIdx.x; state_index < loop_dim; state_index += blockDim.x * gridDim.x) {
        UINT bit_parity = __popcll(state_index & phase_flip_mask)&1;
        int sign = 1 - 2*bit_parity;
        tmp = hipCabs(state[state_index]);
        sum += tmp * tmp * sign;
    }
	sum = warpReduceSum_double(sum);
	if ((threadIdx.x & (warpSize - 1)) == 0){
		atomicAdd_double(&(ret_gpu[0]), sum);
	}
}

__host__ double expectation_value_multi_qubit_Pauli_operator_Z_mask_host(ITYPE phase_flip_mask, void* state, ITYPE dim, void* stream, unsigned int device_number) {
	int current_device = get_current_device();
	if (device_number != current_device) hipSetDevice(device_number);

	GTYPE* state_gpu = reinterpret_cast<GTYPE*>(state);
	hipStream_t* hip_stream = reinterpret_cast<hipStream_t*>(stream);
	hipError_t cudaStatus;
	double ret;
	double* ret_gpu;

	checkCudaErrors(hipMalloc((void**)&ret_gpu, sizeof(double)), __FILE__, __LINE__);
	checkCudaErrors(hipMemsetAsync(ret_gpu, 0, sizeof(double), *hip_stream), __FILE__, __LINE__);

	// this loop_dim is not the same as that of the gpu function
	// and the function uses grid stride loops
	ITYPE loop_dim;

	if (dim <= 64) loop_dim = dim >> 1;
	else if (dim <= (1ULL << 11)) loop_dim = dim >> 2;
	else loop_dim = dim >> 5;

	unsigned int block = loop_dim <= 256 ? loop_dim : 256;
	unsigned int grid = loop_dim / block;

	// unsigned int block = loop_dim <= 1024 ? loop_dim : 1024;
	// unsigned int grid = loop_dim / block;

	expectation_value_multi_qubit_Pauli_operator_Z_mask_gpu << <grid, block, 0, *hip_stream >> > (ret_gpu, phase_flip_mask, state_gpu, dim);

	checkCudaErrors(hipStreamSynchronize(*hip_stream), __FILE__, __LINE__);
	cudaStatus = hipGetLastError();
	checkCudaErrors(cudaStatus, __FILE__, __LINE__);
	checkCudaErrors(hipMemcpyAsync(&ret, ret_gpu, sizeof(double), hipMemcpyDeviceToHost, *hip_stream), __FILE__, __LINE__);
	checkCudaErrors(hipFree(ret_gpu), __FILE__, __LINE__);
	state = reinterpret_cast<void*>(state_gpu);
	// stream = reinterpret_cast<void*>(hip_stream);

	return ret;
}

__host__ double expectation_value_multi_qubit_Pauli_operator_partial_list_host(const UINT* target_qubit_index_list, const UINT* Pauli_operator_type_list, UINT target_qubit_index_count, void* state, ITYPE dim, void* stream, unsigned int device_number) {
	ITYPE bit_flip_mask = 0;
    ITYPE phase_flip_mask = 0;
    UINT global_phase_90rot_count = 0;
    UINT pivot_qubit_index = 0;
    get_Pauli_masks_partial_list_gsim(target_qubit_index_list, Pauli_operator_type_list, target_qubit_index_count,
        &bit_flip_mask, &phase_flip_mask, &global_phase_90rot_count, &pivot_qubit_index);
    double result;
    if(bit_flip_mask == 0){
        result = expectation_value_multi_qubit_Pauli_operator_Z_mask_host(phase_flip_mask, state, dim, stream, device_number);
    }else{
        result = expectation_value_multi_qubit_Pauli_operator_XZ_mask_host(bit_flip_mask, phase_flip_mask, global_phase_90rot_count, pivot_qubit_index, state, dim, stream, device_number);
    }
    return result;
}

__host__ double expectation_value_multi_qubit_Pauli_operator_whole_list_host(const UINT* Pauli_operator_type_list, UINT qubit_count, void* state, ITYPE dim, void* stream, unsigned int device_number) {
	ITYPE bit_flip_mask = 0;
	ITYPE phase_flip_mask = 0;
	UINT global_phase_90rot_count = 0;
	UINT pivot_qubit_index = 0;
	get_Pauli_masks_whole_list_gsim(Pauli_operator_type_list, qubit_count,
		&bit_flip_mask, &phase_flip_mask, &global_phase_90rot_count, &pivot_qubit_index);
	double result;
	if (bit_flip_mask == 0) {
		result = expectation_value_multi_qubit_Pauli_operator_Z_mask_host(phase_flip_mask, state, dim, stream, device_number);
	}
	else {
		result = expectation_value_multi_qubit_Pauli_operator_XZ_mask_host(bit_flip_mask, phase_flip_mask, global_phase_90rot_count, pivot_qubit_index, state, dim, stream, device_number);
	}
	return result;
}

__global__ void transition_amplitude_multi_qubit_Pauli_operator_XZ_mask_gpu(GTYPE* ret_gpu, ITYPE bit_flip_mask, ITYPE phase_flip_mask, UINT global_phase_90rot_count, UINT pivot_qubit_index, const GTYPE* state_bra, const GTYPE* state_ket, ITYPE dim) {
	const ITYPE loop_dim = dim >> 1;

	GTYPE sum = make_hipDoubleComplex(0.0, 0.0);
    GTYPE tmp;
	for (ITYPE state_index = blockIdx.x * blockDim.x + threadIdx.x; state_index < loop_dim; state_index += blockDim.x * gridDim.x) {
		ITYPE basis_0 = insert_zero_to_basis_index_device(state_index, pivot_qubit_index);
		ITYPE basis_1 = basis_0 ^ bit_flip_mask;
		
		UINT sign_0 = __popcll(basis_0 & phase_flip_mask) & 1;
	    tmp = hipCmul( hipCmul(state_ket[basis_0], hipConj(state_bra[basis_1])), PHASE_90ROT_gpu[(global_phase_90rot_count + sign_0 * 2) & 3 ]);
        sum = hipCadd(sum, tmp);

		UINT sign_1 = __popcll(basis_1 & phase_flip_mask) & 1;
		tmp = hipCmul( hipCmul(state_ket[basis_1], hipConj(state_bra[basis_0])), PHASE_90ROT_gpu[(global_phase_90rot_count + sign_1 * 2) & 3]);
        sum = hipCadd(sum, tmp);
	}
	sum.x = warpReduceSum_double(sum.x);
	sum.y = warpReduceSum_double(sum.y);
	if ((threadIdx.x & (warpSize - 1)) == 0){
		atomicAdd_double(&(ret_gpu[0].x), sum.x);
		atomicAdd_double(&(ret_gpu[0].y), sum.y);
	}
}

__host__ CPPCTYPE transition_amplitude_multi_qubit_Pauli_operator_XZ_mask_host(ITYPE bit_flip_mask, ITYPE phase_flip_mask, UINT global_phase_90rot_count, UINT pivot_qubit_index, void* state_bra, void* state_ket, ITYPE dim, void* stream, unsigned int device_number) {
	int current_device = get_current_device();
	if (device_number != current_device) hipSetDevice(device_number);

	GTYPE* state_bra_gpu = reinterpret_cast<GTYPE*>(state_bra);
	GTYPE* state_ket_gpu = reinterpret_cast<GTYPE*>(state_ket);
	hipStream_t* hip_stream = reinterpret_cast<hipStream_t*>(stream);
	CPPCTYPE ret;
	GTYPE* ret_gpu;

	checkCudaErrors(hipMalloc((void**)&ret_gpu, sizeof(GTYPE)), __FILE__, __LINE__);
	checkCudaErrors(hipMemsetAsync(ret_gpu, 0, sizeof(GTYPE), *hip_stream), __FILE__, __LINE__);


	ITYPE loop_dim;
	if (dim <= 32) loop_dim = dim >> 1;
	else if (dim <= 4096) loop_dim = dim >> 2;
	else loop_dim = dim >> 5;

	unsigned int block = loop_dim <= 256 ? loop_dim : 256;
	unsigned int grid = loop_dim / block;

	transition_amplitude_multi_qubit_Pauli_operator_XZ_mask_gpu << <grid, block, 0, *hip_stream >> > (ret_gpu, bit_flip_mask, phase_flip_mask, global_phase_90rot_count, pivot_qubit_index, state_bra_gpu, state_ket_gpu, dim);

	checkCudaErrors(hipStreamSynchronize(*hip_stream), __FILE__, __LINE__);
	checkCudaErrors(hipGetLastError(), __FILE__, __LINE__);
	checkCudaErrors(hipMemcpyAsync(&ret, ret_gpu, sizeof(GTYPE), hipMemcpyDeviceToHost, *hip_stream), __FILE__, __LINE__);
	checkCudaErrors(hipFree(ret_gpu), __FILE__, __LINE__);
	state_bra = reinterpret_cast<void*>(state_bra_gpu);
	state_ket = reinterpret_cast<void*>(state_ket_gpu);
	// stream = reinterpret_cast<void*>(hip_stream);
	return ret;
}

__global__ void transition_amplitude_multi_qubit_Pauli_operator_Z_mask_gpu(GTYPE* ret, ITYPE phase_flip_mask, GTYPE* state_bra, GTYPE* state_ket, ITYPE dim) {
	const ITYPE loop_dim = dim;
	GTYPE sum = make_hipDoubleComplex(0.0, 0.0);
	for (ITYPE state_index = blockIdx.x * blockDim.x + threadIdx.x; state_index < loop_dim; state_index += blockDim.x * gridDim.x) {
		int bit_parity = __popcll(state_index & phase_flip_mask) & 1;
		int sign = 1 - 2 * bit_parity;
		GTYPE tmp = hipCmul( state_ket[state_index], hipConj(state_bra[state_index]));
		tmp = hipCmul( make_hipDoubleComplex( (double)sign,0.0), tmp);
        sum = hipCadd(sum, tmp);
    }
	sum.x = warpReduceSum_double(sum.x);
	sum.y = warpReduceSum_double(sum.y);

	if ((threadIdx.x & (warpSize - 1)) == 0){
		atomicAdd_double(&(ret[0].x), sum.x);
		atomicAdd_double(&(ret[0].y), sum.y);
	}
}

__host__ CPPCTYPE transition_amplitude_multi_qubit_Pauli_operator_Z_mask_host(ITYPE phase_flip_mask, void* state_bra, void* state_ket, ITYPE dim, void* stream, unsigned int device_number) {
	int current_device = get_current_device();
	if (device_number != current_device) hipSetDevice(device_number);

	hipError_t cudaStatus;
	GTYPE* state_bra_gpu = reinterpret_cast<GTYPE*>(state_bra);
	GTYPE* state_ket_gpu = reinterpret_cast<GTYPE*>(state_ket);
	hipStream_t* hip_stream = reinterpret_cast<hipStream_t*>(stream);
	CPPCTYPE ret;
	GTYPE* ret_gpu;

	checkCudaErrors(hipMalloc((void**)&ret_gpu, sizeof(GTYPE)), __FILE__, __LINE__);
	checkCudaErrors(hipMemsetAsync(ret_gpu, 0, sizeof(GTYPE), *hip_stream), __FILE__, __LINE__);

	ITYPE loop_dim;
	if (dim <= 32) loop_dim = dim >> 1;
	else if (dim <= 4096) loop_dim = dim >> 2;
	else loop_dim = dim >> 5;

	unsigned int block = loop_dim <= 256 ? loop_dim : 256;
	unsigned int grid = loop_dim / block;

	transition_amplitude_multi_qubit_Pauli_operator_Z_mask_gpu << <grid, block, 0, *hip_stream >> > (ret_gpu, phase_flip_mask, state_bra_gpu, state_ket_gpu, dim);

	checkCudaErrors(hipStreamSynchronize(*hip_stream), __FILE__, __LINE__);
	cudaStatus = hipGetLastError();
	checkCudaErrors(cudaStatus, __FILE__, __LINE__);
	checkCudaErrors(hipMemcpyAsync(&ret, ret_gpu, sizeof(GTYPE), hipMemcpyDeviceToHost, *hip_stream), __FILE__, __LINE__);
	checkCudaErrors(hipFree(ret_gpu), __FILE__, __LINE__);
	state_bra = reinterpret_cast<void*>(state_bra_gpu);
	state_ket = reinterpret_cast<void*>(state_ket_gpu);
	// stream = reinterpret_cast<void*>(hip_stream);
	return ret;
}

__host__ CPPCTYPE transition_amplitude_multi_qubit_Pauli_operator_partial_list_host(const UINT* target_qubit_index_list, const UINT* Pauli_operator_type_list, UINT target_qubit_index_count, void* state_bra, void* state_ket, ITYPE dim, void* stream, unsigned int device_number) {
	ITYPE bit_flip_mask = 0;
	ITYPE phase_flip_mask = 0;
	UINT global_phase_90rot_count = 0;
	UINT pivot_qubit_index = 0;
	get_Pauli_masks_partial_list_gsim(target_qubit_index_list, Pauli_operator_type_list, target_qubit_index_count,
		&bit_flip_mask, &phase_flip_mask, &global_phase_90rot_count, &pivot_qubit_index);
	CPPCTYPE result;
	if (bit_flip_mask == 0) {
		result = transition_amplitude_multi_qubit_Pauli_operator_Z_mask_host(phase_flip_mask, state_bra, state_ket, dim, stream, device_number);
	}
	else {
		result = transition_amplitude_multi_qubit_Pauli_operator_XZ_mask_host(bit_flip_mask, phase_flip_mask, global_phase_90rot_count, pivot_qubit_index, state_bra, state_ket, dim, stream, device_number);
	}
	return result;
}

__host__ CPPCTYPE transition_amplitude_multi_qubit_Pauli_operator_whole_list_host(const UINT* Pauli_operator_type_list, UINT qubit_count, void* state_bra, void* state_ket, ITYPE dim, void* stream, unsigned int device_number) {
	ITYPE bit_flip_mask = 0;
	ITYPE phase_flip_mask = 0;
	UINT global_phase_90rot_count = 0;
	UINT pivot_qubit_index = 0;
	get_Pauli_masks_whole_list_gsim(Pauli_operator_type_list, qubit_count,
		&bit_flip_mask, &phase_flip_mask, &global_phase_90rot_count, &pivot_qubit_index);
	CPPCTYPE result;
	if (bit_flip_mask == 0) {
		result = transition_amplitude_multi_qubit_Pauli_operator_Z_mask_host(phase_flip_mask, state_bra, state_ket, dim, stream, device_number);
	}
	else {
		result = transition_amplitude_multi_qubit_Pauli_operator_XZ_mask_host(bit_flip_mask, phase_flip_mask, global_phase_90rot_count, pivot_qubit_index, state_bra, state_ket, dim, stream, device_number);
	}
	return result;
}
